#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include ""
#include <time.h>
#include <math.h>

// ��ʱ
clock_t start, stop; 
double duration;
size_t avail, total;

#ifdef __linux__
 #define  CLK_TCK CLOCKS_PER_SEC
#endif

dim3 threadsPerBlock(32, 32);
size_t maxThreads;
int max_gpu_index = 0;


//��ʼ�����㺯��
__global__ void set_Zero(hipfftComplex* cuda_result, int Height, int Width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < Height && col < Width)
	{
		cuda_result[row * Width + col].x = 0;
		cuda_result[row * Width + col].y = 0;
	}
}


//��w�����ʼ����ֵ����
__global__ void initKernel_W(hipfftComplex* devPitchedPtr, size_t pitch, int Height, int Width, int Depth, int VOL,
                             double* init_real, double* init_img)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int swi_id = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;

	if (swi_id < Depth)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)devPitchedPtr + swi_id * pitch);
		if (col < Width && row < Height)
		{
			rowHead[row * Width + col].x = (float)init_real[row * Width + col];
			rowHead[row * Width + col].y = (float)init_img[row * Width + col];
		}
	}
}

//��֯cal_atom_ion_part1��exp_kxX_kyY1��sum_exp_prodִ��˳����
__global__ void cal_atomORion_fun(hipfftComplex* cuda_result, float* coff, double* s_2, int atom_nums
                                  , double* kx, double* ky, int Height, int Width, int VOL)
{
	//���������Ի�ԭ����
	//cal_atom_ion_part1�����Ľ�������sum��
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atom_nums)
	{
		if (row < Height && col < Width)
		{
			float part1 = coff[atom * 14 + 3] * (2 + coff[atom * 14 + 4] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 4] * s_2[row * Width + col] * 4, 2);
			float part2 = coff[atom * 14 + 5] * (2 + coff[atom * 14 + 6] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 6] * s_2[row * Width + col] * 4, 2);
			float part3 = coff[atom * 14 + 7] * (2 + coff[atom * 14 + 8] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 8] * s_2[row * Width + col] * 4, 2);
			float part4 = coff[atom * 14 + 9] * (2 + coff[atom * 14 + 10] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 10] * s_2[row * Width + col] * 4, 2);
			float part5 = coff[atom * 14 + 11] * (2 + coff[atom * 14 + 12] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 12] * s_2[row * Width + col] * 4, 2);
			float sum = coff[atom * 14 + 2] * (part1 + part2 + part3 + part4 + part5) * expf((-1) * coff[atom * 14 + 13] * s_2[row * Width + col]);
			float temp = (-2) * (coff[atom * 14 + 0] * kx[row * Width + col] + coff[atom * 14 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + cosf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + sinf(temp) * sum;
		}
	}
}

__global__ void cal_absorb_fun(hipfftComplex* cuda_result, float* coff, float* coff_absob, double* s_2, int atom_nums
                               , double* kx, double* ky, int Height, int Width, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atom_nums)
	{
		if (row < Height && col < Width)
		{
			
			float part1 = coff_absob[atom * 10 + 0] * expf((-1) * coff_absob[atom * 10 + 1] * s_2[row * Width + col]);
			float part2 = coff_absob[atom * 10 + 2] * expf((-1) * coff_absob[atom * 10 + 3] * s_2[row * Width + col]);
			float part3 = coff_absob[atom * 10 + 4] * expf((-1) * coff_absob[atom * 10 + 5] * s_2[row * Width + col]);
			float part4 = coff_absob[atom * 10 + 6] * expf((-1) * coff_absob[atom * 10 + 7] * s_2[row * Width + col]);
			float part5 = coff_absob[atom * 10 + 8] * expf((-1) * coff_absob[atom * 10 + 9] * s_2[row * Width + col]);
			float sum = coff[atom * 14 + 2] * (part1 + part2 + part3 + part4 + part5);
			float temp = (-2) * (coff[atom * 14 + 0] * kx[row * Width + col] + coff[atom * 14 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + (-1) * sinf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + cosf(temp) * sum;
		
		}
	}
}


//һ������Ԫ��ԭ���Ի������Լ��϶�Ӧ������ֵ�����������cuda_result��
__global__ void add_fun_absob(hipfftComplex* cuda_result, hipfftComplex* cuda_absorb, int Height, int Width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < Height && col < Width)
	{
		cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + cuda_absorb[row * Width + col].x;
		cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + cuda_absorb[row * Width + col].y;
	}
}


__global__ void cal_atomAndion_fun(hipfftComplex* cuda_result,float* coff, double* s_2, int atomi_nums
                                   , double* kx, double* ky
                                   , int Height, int Width, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atomi_nums)
	{
		if (row < Height && col < Width)
		{
			float atom_part1 = coff[atom * 26 + 4] * (2 + coff[atom * 26 + 5] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 5] * s_2[row * Width + col] * 4, 2);
			float atom_part2 = coff[atom * 26 + 6] * (2 + coff[atom * 26 + 7] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 7] * s_2[row * Width + col] * 4, 2);
			float atom_part3 = coff[atom * 26 + 8] * (2 + coff[atom * 26 + 9] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 9] * s_2[row * Width + col] * 4, 2);
			float atom_part4 = coff[atom * 26 + 10] * (2 + coff[atom * 26 + 11] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 11] * s_2[row * Width + col] * 4, 2);
			float atom_part5 = coff[atom * 26 + 12] * (2 + coff[atom * 26 + 13] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 13] * s_2[row * Width + col] * 4, 2);
			float atom_sum = coff[atom * 26 + 3] * (atom_part1 + atom_part2 + atom_part3 + atom_part4 + atom_part5);

			float ion_part1 = coff[atom * 26 + 15] * (2 + coff[atom * 26 + 16] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 16] * s_2[row * Width + col] * 4, 2);
			float ion_part2 = coff[atom * 26 + 17] * (2 + coff[atom * 26 + 18] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 18] * s_2[row * Width + col] * 4, 2);
			float ion_part3 = coff[atom * 26 + 19] * (2 + coff[atom * 26 + 20] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 20] * s_2[row * Width + col] * 4, 2);
			float ion_part4 = coff[atom * 26 + 21] * (2 + coff[atom * 26 + 22] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 22] * s_2[row * Width + col] * 4, 2);
			float ion_part5 = coff[atom * 26 + 23] * (2 + coff[atom * 26 + 24] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 24] * s_2[row * Width + col] * 4, 2);
			float ion_sum = coff[atom * 26 + 14] * (ion_part1 + ion_part2 + ion_part3 + ion_part4 + ion_part5);

			float sum = coff[atom * 26 + 2] * (atom_sum + ion_sum) * expf((-1) * coff[atom * 26 + 25] * s_2[row * Width + col]);

			float temp = (-2) * (coff[atom * 26 + 0] * kx[row * Width + col] + coff[atom * 26 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + cosf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + sinf(temp) * sum;
			
		}
	}	
}

__global__ void cal_absorb_fun2(hipfftComplex* cuda_result, float* coff, float* coff_absob, double* s_2, int atomi_nums
                                , double* kx, double* ky
                                , int Height, int Width, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atomi_nums)
	{
		if (row < Height && col < Width)
		{	
			float part1 = coff_absob[atom * 10 + 0] * expf((-1) * coff_absob[atom * 10 + 1] * s_2[row * Width + col]);
			float part2 = coff_absob[atom * 10 + 2] * expf((-1) * coff_absob[atom * 10 + 3] * s_2[row * Width + col]);
			float part3 = coff_absob[atom * 10 + 4] * expf((-1) * coff_absob[atom * 10 + 5] * s_2[row * Width + col]);
			float part4 = coff_absob[atom * 10 + 6] * expf((-1) * coff_absob[atom * 10 + 7] * s_2[row * Width + col]);
			float part5 = coff_absob[atom * 10 + 8] * expf((-1) * coff_absob[atom * 10 + 9] * s_2[row * Width + col]);
			float sum = coff[atom * 26 + 2] * (part1 + part2 + part3 + part4 + part5);
			float temp = (-2) * (coff[atom * 26 + 0] * kx[row * Width + col] + coff[atom * 26 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + (-1) * sinf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + cosf(temp) * sum;
		}
	}
}


__global__ void copy_result_to_p(hipfftComplex* result, hipfftComplex* P, int Height, int Width, float parameter)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < Height && col < Width)
	{
		P[row * Width + col].x = result[row * Width + col].x * parameter + P[row * Width + col].x;
		P[row * Width + col].y = result[row * Width + col].y * parameter + P[row * Width + col].y;
	}
}

//��P����ifftShift����
__global__ void ifftShift(hipfftComplex* P_PitchedPtr, size_t pitch, int P_Height, int P_Width, int P_Slices, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int d_id = id / (VOL / 2);
	int b_id = id % (VOL / 2) / (VOL / 4);
	int e_id = id % (VOL / 2) % (VOL / 4);

	int row = e_id / (P_Width / 2);
	int col = e_id % (P_Width / 2);

	int dest_row, dest_col;

	if (d_id < P_Slices)
	{
		if (b_id == 0)
		{
			// if(row == 0 && col == 0 && d_id == 0)
			// 	printf("invoke");
			dest_row = row + (P_Height / 2);
			dest_col = col + (P_Width / 2);
		}
		else if (b_id == 1)
		{
			col = col + (P_Width / 2);
			dest_row = row + (P_Height / 2);
			dest_col = col - (P_Width / 2);
		}
		// if(row == 0 && col == 2 && d_id == 0)
		// 	printf("(%d,%d,%d),(%d,%d),(%d,%d)\n",b_id,row,col,dest_row,dest_col,P_Height,P_Width);
		hipfftComplex* rowHead = (hipfftComplex *)((char *)P_PitchedPtr + d_id * pitch);
		float e1_real = rowHead[row * P_Width + col].x;
		float e1_img = rowHead[row * P_Width + col].y;

		float e2_real = rowHead[dest_row * P_Width + dest_col].x;
		float e2_img = rowHead[dest_row * P_Width + dest_col].y;

		rowHead[row * P_Width + col].x = e2_real;
		rowHead[row * P_Width + col].y = e2_img;

		rowHead[dest_row * P_Width + dest_col].x = e1_real;
		rowHead[dest_row * P_Width + dest_col].y = e1_img;
	}
}

//p��AperTrue������
__global__ void p_aperTrue_pointMul(hipfftComplex* P_PitchedPtr, size_t pitch, double* aperTrue, int P_Height,
                                    int P_Width, int Slices, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int p_id = id / VOL;
	int row = (id % VOL) / P_Width;
	int col = (id % VOL) % P_Width;

	if (p_id < Slices)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)P_PitchedPtr + p_id * pitch);

		if (col < P_Width && row < P_Height)
		{
			rowHead[col + row * P_Width].x = rowHead[col + row * P_Width].x * aperTrue[col + row * P_Width];
			rowHead[col + row * P_Width].y = rowHead[col + row * P_Width].y * aperTrue[col + row * P_Width];
		}
	}
}

//�Խ���ifft2��ľ�������һ����������
__global__ void scaler(hipfftComplex* devPitchedPtr, size_t pitch, int Height, int Width, int Depth, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int swi_id = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;

	if (swi_id < Depth)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)devPitchedPtr + swi_id * pitch);

		if (col < Width && row < Height)
		{
			rowHead[col + row * Width].x = rowHead[col + row * Width].x / (Height * Width);
			rowHead[col + row * Width].y = rowHead[col + row * Width].y / (Height * Width);
		}
	}
}


//����exp(i * simga * P)
__global__ void p_exp_sigma(hipfftComplex* P_PitchedPtr, size_t pitch, int P_Height, int P_Width, int Slices, int VOL,
                            float sigma)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int p_id = id / VOL;
	int row = (id % VOL) / P_Width;
	int col = (id % VOL) % P_Width;

	if (p_id < Slices)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)P_PitchedPtr + p_id * pitch);

		if (col < P_Width && row < P_Height)
		{
			float real = (-1) * rowHead[col + row * P_Width].y * sigma;
			float img = rowHead[col + row * P_Width].x * sigma;

			rowHead[col + row * P_Width].x = expf(real) * cosf(img);
			rowHead[col + row * P_Width].y = expf(real) * sinf(img);
		}
	}
}

//��proj�����˲�������
__global__ void w_p_pointMul(hipfftComplex* W, hipfftComplex* P, size_t pitch1
                             , int Height, int Width, int w_Depth, int VOL, int P_Width
                             , int beginRow, int beginCol
                             , int width_red, int step, int start)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	//��λ���߳����ڵڼ���w��
	int swi_id = id / VOL + start;
	//��ÿ��w�ڲ���λ���ǵڼ���
	int row = (id % VOL) / Width;
	//��ÿ��w�ڲ���λ���ǵڼ���
	int col = (id % VOL) % Width;
	int a = swi_id - start;
	if (a < w_Depth)
	{
		//��ȡ��ÿһ�����W��ά������׵�ַ
		hipfftComplex* rowHead1 = (hipfftComplex *)((char *)W + a * pitch1);
		// ӳ�䵽��p�����swi_id��Сp���׵�ַ
		int x_mat = beginRow + (swi_id / width_red) * step;
		int y_mat = beginCol + (swi_id % width_red) * step;
		//���е�˲���
		float temp;
		if (col < Width && row < Height)
		{
			temp = rowHead1[row * Width + col].x;
			//Сp��w�������Ԫ����P������
			int p_e_x = x_mat + row;
			int p_e_y = y_mat + col;
			//Сp��w�������Ԫ����P��һά�洢ʱ���±�
			int p_e = p_e_x * P_Width + p_e_y;
			int w_e = row * Width + col;

			rowHead1[w_e].x = temp * P[p_e].x - rowHead1[w_e].y * P[p_e].y;
			rowHead1[w_e].y = temp * P[p_e].y + rowHead1[w_e].y * P[p_e].x;
		}
	}
}

//����մ��������˲�������
__global__ void w_constM_pointMul(hipfftComplex* devPitchedPtr, size_t pitch, hipfftComplex* constMat, int Height,
                                  int Width, int Depth, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int swi_id = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if (swi_id < Depth)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)devPitchedPtr + swi_id * pitch);
		float temp;
		if (col < Width && row < Height)
		{
			// if (row==0 && col==0 && swi_id == 0)
			// {
			// 	printf("row:%d - col:%d\n",row,col);
			// 	printf("rowHead.x:%0.5lf + rowHead.y:%0.5f\n",rowHead[row * Width + col].x,rowHead[row * Width + col].y);
			// 	printf("constMat.x:%0.5lf + constMat.y:%0.5f\n",constMat[col + row * Width].x,constMat[row * Width + col].y);
			// }
			temp = rowHead[col + row * Width].x;
			rowHead[col + row * Width].x = temp * constMat[col + row * Width].x - rowHead[col + row * Width].y *
				constMat[col + row * Width].y;
			rowHead[col + row * Width].y = temp * constMat[col + row * Width].y + rowHead[col + row * Width].y *
				constMat[col + row * Width].x;
			//printf("rowHead.x:%0.5lf + rowHead.y:%0.5f\n",rowHead[row * Width + col].x,rowHead[row * Width + col].y);
		}
	}
}

__global__ void abs_w_final(hipfftComplex* devPitchedPtr, double* aper_result, size_t pitch, size_t p, int Height,
                            int Width, int Depth, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int swi_id = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;

	if (swi_id < Depth)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)devPitchedPtr + swi_id * pitch);
		double* rHead = (double *)((char *)aper_result + swi_id * p);
		if (col < Width && row < Height)
		{
			double temp1 = rowHead[col + row * Width].x;
			double temp2 = rowHead[col + row * Width].y;
			//hypotfΪ��������������ƽ���͵�ƽ����
			rHead[col + row * Width] =  powf( hypotf(temp1, temp2), 2);
			// rowHead[col + row * Width].x = hypotf(temp1, temp2);
			// rowHead[col + row * Width].y = 0;
		}
	}
}


// w_final������aper������е��
__global__ void w_aper_pointMul(double* result, double* aper, size_t pitch, int Height, int Width, int Depth, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int swi_id = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if (swi_id < Depth)
	{
		double* rowHead = (double *)((char *)result + swi_id * pitch);
		double temp;
		if (col < Width && row < Height)
		{
			temp = rowHead[col + row * Width];
			rowHead[col + row * Width] = temp * aper[col + row * Width];
		}
	}
}


__global__ void absw_and_aperpm(hipfftComplex* devPitchedPtr, double* aper_result, double* aper, size_t pitch, size_t p, int Height,
                            int Width, int Depth, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int swi_id = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;

	if (swi_id < Depth)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)devPitchedPtr + swi_id * pitch);
		double* rHead = (double *)((char *)aper_result + swi_id * p);
		if (col < Width && row < Height)
		{
			double temp1 = rowHead[col + row * Width].x;
			double temp2 = rowHead[col + row * Width].y;
			//hypotfΪ��������������ƽ���͵�ƽ����
			rHead[col + row * Width] =  powf( hypotf(temp1, temp2), 2) * aper[col + row * Width];
		}
	}
}


//W_final��aper����Ժ�ľ�����ͽ�������result_sum_dev_head��
__global__ void sum_w_kernel(double* aper_result, size_t pitch
							 , int result_size, int thread_size, 
							 double* result_sum_dev_head, int Depth)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + 
	(threadIdx.y * blockDim.x + threadIdx.x);
	int result_id = id / thread_size; //�����߳�����thread_size*Depth��thread_size�Ǵ���һ��result�����Լ�������߳���
	int r_e = id % thread_size; //r_e����ǰ�߳�Ҫ����ĵ�swi_id��w��Ԫ��
	if (r_e + thread_size > result_size)
	{
		//�����Ѿ�����w����Ԫ�صĸ�����
		return;
	}
	double* result_rowHead = (double *)((char *)aper_result + result_id * pitch);
	if (result_id < Depth)
	{
		result_rowHead[r_e] = result_rowHead[r_e] + result_rowHead[r_e + thread_size];
		//printf("%d,%d,%lf\n",r_e,id,result_rowHead[r_e]);
	}
	if (thread_size == 1)
	{
		//����ÿ��result����ֻʣ���������Ԫ�ص���ͣ�����ֻ��Ҫһ���̣߳���thread_size = 1��
		//����ʵ�ʺ˺�������Depth���߳�ȥ����Depth��result���󣬴�ʱid��Ӧ�ľ��ǵڼ���result
		// printf("%d,%d,%lf\n",r_e,id,result_rowHead[0]);
		result_sum_dev_head[id] = result_rowHead[0];
	}
}


void printf_2d(hipfftComplex* w_PitchedPtr, int width, int height, int depth, size_t pitch)
{
	//w_s��������ɵ���ά������������ڴ���׵�ַָ��
	hipfftComplex* w_p = (hipfftComplex *)malloc(sizeof(hipfftComplex) * width * height * depth);
	hipMemcpy2D(w_p, width * height * sizeof(hipfftComplex), w_PitchedPtr, pitch,
	             width * height * sizeof(hipfftComplex), depth, hipMemcpyDeviceToHost);
	//printf("%.12lf + (%.12lf)i\n", w_p[127*256+127].x, w_p[127*256+127].y);
	int k = 0;
	//for (int i = 496 * 235 + 273; i < 496 * 235 + 273 + 5; i++)
	for (int i = 0; i < 5; i++)
	{
		printf("%d================%.12lf + (%.12lf)i \n", i, w_p[i].x, w_p[i].y);
		k++;
		if ((k % (height * width)) == 0)
		{
			printf("\n=======================\n");
			printf("\n");
		}
	}
	if (w_p!=NULL)
	{
		free(w_p);
	}
}

void printf_1d(hipfftComplex* cuda_result, int height, int width)
{
	hipfftComplex* test = (hipfftComplex *)malloc(sizeof(hipfftComplex) * height * width);
	hipMemcpy(test, cuda_result, height * sizeof(hipfftComplex) * width, hipMemcpyDeviceToHost);
	int k = 0;
	for (int i = 0; i < height * width; i++)
	{
		k++;
		printf("%lf + (%lf)i ", test[i].x, test[i].y);
		if (k % width == 0)
		{
			printf("\n");
		}
	}
	if (test!=NULL)
	{
		free(test);
	}
}

void printf_dd(double* cuda_result, int start, int len)
{
	double* test = (double *)malloc(sizeof(double) * len);
	hipMemcpy(test, cuda_result+start, sizeof(double) * len, hipMemcpyDeviceToHost);
	for (int i = 0; i < len; i++)
	{
		if (fabs(test[i]-0) > 1E-6){
			printf("printf_dd[%d]: %lf\n", start+i, test[i]);
		}
	}
	if (test!=NULL)
	{
		free(test);
	}
}



void setThreadGrid(int nums, int VOL, int* blockX, int* blockY)
{
	int blockThread = threadsPerBlock.x * threadsPerBlock.y;
	int blockSum = (nums * VOL  + blockThread - 1) / blockThread;
	if (blockSum < blockThread)
	{
		*blockX = 1;
		*blockY = blockSum;
	}
	else
	{
		*blockX = (blockSum + blockThread - 1) / blockThread;
		*blockY = (blockSum + *blockX - 1) / *blockX;
	}
}

size_t get_memory(int gpu_index)
{
	size_t gpu_free = 0;
	size_t gpu_total = 0;
	hipSetDevice(gpu_index);
	hipMemGetInfo(&gpu_free, &gpu_total);
	return gpu_free;
}

// �������е�p
void cal_p(double* atom_slice, int slice, double* absorb_n, int* sumSeries_n, double* proj_coff_mat, dim3 blockSize_P,
            hipfftComplex* cuda_result1, int p_height, int p_width,
           double* atom_slice_i, double* absorb_n_i, double* proj_coff_mat_i,
           hipfftComplex* cuda_result2, double* cuda_s_2, int* sumSeries_n_i,
           hipfftComplex* p_PitchedPtr, size_t pitch2, double* cuda_kx, double* cuda_ky,
           double parameter, int *allAtoms, int *allAtomsi)
{
	//������������Ժ�ԭ���Ծ�������Ҫ�Ĳ�������
	float* coffMat;
	float* coffAbsob;
	float* coffMat_cuda;
	float* coffAbsob_cuda;

	//cuda_result_absorb���濼������
	hipfftComplex* cuda_result_absorb;
	//һ���н��������Ի�ԭ����Ԫ�صĵ���
	// slice��1��ʼ
	int atom_nums = atom_slice[slice - 1];

	size_t ava = get_memory(max_gpu_index);
	printf("%d-------%d-------%zd MB\n",slice, atom_nums, ava/1024/1024);
	*allAtoms += atom_nums;
	int p_vol = p_width * p_height;
	int x, y;

	int p_batchs = maxThreads/p_vol;
	// printf("p_batchs: %d\n", p_batchs);
	if (atom_nums > 0)
	{
		int ci = atom_nums / p_batchs + 1;
		if (absorb_n == NULL)
		{
			int len = 14 * atom_nums;
			coffMat = (float *)malloc(sizeof(float) * len);
			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atom_nums; m++)
			{
				for (int n = 0; n < 14; n++)
					coffMat[m * 14 + n] = proj_coff_mat[*sumSeries_n * 14 + m * 14 + n];
			}
			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len, hipMemcpyHostToDevice);
			set_Zero << < blockSize_P, threadsPerBlock >> >(cuda_result1, p_height, p_width);
			//�����Ѵ��ڴ渴�Ƶ��Դ��У����ͷ����ڴ��п��ٵ��ڴ档
			if (coffMat!=NULL)
			{
				free(coffMat);
			}
			
			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atom_nums)
				{
					end_atom = atom_nums;
				}
				int batch_atom = end_atom - start_atom;
				// printf("start_atom:%d end_atom:%d batch_atom:%d\n",start_atom, end_atom, batch_atom);
				if(batch_atom > 0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
					cal_atomORion_fun << < grid, threadsPerBlock >> >(cuda_result1, coffMat_cuda+start_atom*14, cuda_s_2, batch_atom
																			, cuda_kx, cuda_ky, p_height, p_width, p_vol);
					hipDeviceSynchronize();
				}
			}
		}
		else
		{
			hipMalloc((void**)&cuda_result_absorb, sizeof(hipfftComplex) * p_height * p_width);
			//����ò�ϵ������Ҫ�Ŀռ䳤��
			int len1 = 14 * atom_nums;
			int len2 = 10 * atom_nums;
			coffMat = (float *)malloc(sizeof(float) * len1);
			coffAbsob = (float *)malloc(sizeof(float) * len2);

			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atom_nums; m++)
			{
				for (int n = 0; n < 14; n++)
					coffMat[m * 14 + n] = proj_coff_mat[*sumSeries_n * 14 + m * 14 + n];
				for (int n = 0; n < 10; n++)
					coffAbsob[m * 10 + n] = absorb_n[*sumSeries_n * 10 + m * 10 + n];
			}
			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len1);
			hipMalloc((void**)&coffAbsob_cuda, sizeof(float) * len2);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len1, hipMemcpyHostToDevice);
			hipMemcpy(coffAbsob_cuda, coffAbsob, sizeof(float) * len2, hipMemcpyHostToDevice);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result1, p_height, p_width);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result_absorb, p_height, p_width);
			//�����Ѵ��ڴ渴�Ƶ��Դ��У����ͷ����ڴ��п��ٵ��ڴ档
			if (coffMat!=NULL)
			{
				free(coffMat);
			}
			if (coffAbsob!=NULL)
			{
				free(coffAbsob);
			}
			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atom_nums)
				{
					end_atom = atom_nums;
				}
				int batch_atom = end_atom - start_atom;
				if(batch_atom>0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
					// ���е�silce��͸��ʱ��cal_atomORion_fun����ĳ��Ԫ�ص� occupy_rate * ��i(1-5)Ai*exp(-Bi * s^2).* exp(-(kx*X + ky*Y)*i)
					// ���㲻���յ�һ����
					cal_atomORion_fun << <grid, threadsPerBlock >> >(cuda_result1, coffMat_cuda + start_atom * 14, cuda_s_2, batch_atom
																				, cuda_kx, cuda_ky, p_height, p_width, p_vol);
																				hipDeviceSynchronize();
																				//cal_absorb_fun ���� occupy_rate * ��i(1-5)j * Ci * exp(-Di * s^2).* exp(-(kx*X + ky*Y)*i)
					cal_absorb_fun << <grid, threadsPerBlock >> >(cuda_result_absorb, coffMat_cuda + start_atom * 14, coffAbsob_cuda + start_atom * 10
						, cuda_s_2, batch_atom, cuda_kx, cuda_ky, p_height, p_width, p_vol);
						hipDeviceSynchronize();
				}
			}		
		}
	}
	//�������գ���������������������
	if ((absorb_n != NULL) && (atom_nums != 0))
	{
		add_fun_absob << <blockSize_P, threadsPerBlock >> >(cuda_result1, cuda_result_absorb, p_height, p_width);
		hipDeviceSynchronize();
		//�������գ����ͷ���Ӧ���Դ�
		if (cuda_result_absorb!=NULL)
		{
			hipFree(cuda_result_absorb);
		}
		
	}
	//�ͷŵ�ǰ�����Դ濪�ٵĿռ�
				
	if(coffMat_cuda!=NULL)
	{
		hipFree(coffMat_cuda);
	}
	if (coffAbsob_cuda!=NULL)
	{
		hipFree(coffAbsob_cuda);
	}
	//�ۼ�һ����ƫ��ֵ
	*sumSeries_n += atom_nums;
	int atomi_nums = atom_slice_i[slice - 1];
	
	allAtomsi += atomi_nums;
	//һ�������м���ԭ��������������Ԫ�صĵ���
	if (atomi_nums > 0)
	{
		int ci = atomi_nums / p_batchs + 1;
		if (absorb_n_i == NULL)
		{
			//��proj_coff_mat_i�л�ȡ����������Ҫ������ϵ��
			//����ò�ϵ������Ҫ�Ŀռ䳤��
			int len = 26 * atomi_nums;
			coffMat = (float *)malloc(sizeof(float) * len);
			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atomi_nums; m++)
			{
				for (int n = 0; n < 26; n++)
					coffMat[m * 26 + n] = proj_coff_mat_i[*sumSeries_n_i * 26 + m * 26 + n];
			}
			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len, hipMemcpyHostToDevice);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result2, p_height, p_width);
			if (coffMat!=NULL)
			{
				free(coffMat);
			}

			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atomi_nums)
				{
					end_atom = atomi_nums;
				}
				int batch_atom = end_atom - start_atom;
				if(batch_atom>0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
					cal_atomAndion_fun << <grid, threadsPerBlock >> >(cuda_result2, coffMat_cuda + start_atom * 26, cuda_s_2, batch_atom
																		, cuda_kx, cuda_ky, p_height, p_width, p_vol);
																		hipDeviceSynchronize();
				}
			}
		}
		else
		{
			//��proj_coff_mat��absorp�л�ȡ�����м�������Ҫ������ϵ��
			hipMalloc((void**)&cuda_result_absorb, sizeof(hipfftComplex) * p_height * p_width);

			//����ò�ϵ������Ҫ�Ŀռ䳤��
			int len1 = 26 * atomi_nums;
			int len2 = 10 * atomi_nums;

			coffMat = (float *)malloc(sizeof(float) * len1);
			coffAbsob = (float *)malloc(sizeof(float) * len2);

			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atomi_nums; m++)
			{
				for (int n = 0; n < 26; n++)
					coffMat[m * 26 + n] = proj_coff_mat_i[*sumSeries_n_i * 26 + m * 26 + n];
				for (int n = 0; n < 10; n++)
					coffAbsob[m * 10 + n] = absorb_n_i[*sumSeries_n_i * 10 + m * 10 + n];
			}

			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len1);
			hipMalloc((void**)&coffAbsob_cuda, sizeof(float) * len2);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len1, hipMemcpyHostToDevice);
			hipMemcpy(coffAbsob_cuda, coffAbsob, sizeof(float) * len2, hipMemcpyHostToDevice);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result2, p_height, p_width);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result_absorb, p_height, p_width);
			//�����Ѵ��ڴ渴�Ƶ��Դ��У����ͷ����ڴ��п��ٵ��ڴ档
			if (coffMat!=NULL)
			{
				free(coffMat);
			}
			if(coffAbsob!=NULL)
			{
				free(coffAbsob);
			}

			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atomi_nums)
				{
					end_atom = atomi_nums;
				}
				int batch_atom = end_atom - start_atom;
				if(batch_atom>0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
				
					cal_atomAndion_fun << <grid, threadsPerBlock >> >(
							cuda_result2, coffMat_cuda + start_atom * 26, cuda_s_2, batch_atom
						, cuda_kx, cuda_ky
						, p_height, p_width, p_vol);
						hipDeviceSynchronize();
					cal_absorb_fun2 << <grid, threadsPerBlock >> >(
						cuda_result_absorb, coffMat_cuda + start_atom * 26, coffAbsob_cuda + start_atom * 10, cuda_s_2, batch_atom
						, cuda_kx, cuda_ky
						, p_height, p_width, p_vol);	
						hipDeviceSynchronize();
				}
			}
		}
	}
	
	//�����������
	//��������������ԭ���Ե�Ԫ���������������
	if ((absorb_n_i != NULL) && (atomi_nums != 0))
	{
		add_fun_absob << <blockSize_P, threadsPerBlock >> >(cuda_result2, cuda_result_absorb, p_height,
		                                                      p_width);
		//�������ղ��ͷ���Ӧ���Դ�
		if(cuda_result_absorb!=NULL)
		{
			hipFree(cuda_result_absorb);
		}
		
	}
	//�ͷŵ�ǰ�����Դ濪�ٵĿռ�
	if (coffMat_cuda!=NULL)
	{
		hipFree(coffMat_cuda);
	}
	if(coffAbsob_cuda!=NULL)
	{
		hipFree(coffAbsob_cuda);
	}
	//�ۼƵ�ǰ����ƫ����
	*sumSeries_n_i += atomi_nums;

	//��һ�����н������ӻ�ԭ���Ը�������������ԭ���Ե�Ԫ�����
	if (atom_nums == 0)
		set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result1, p_height, p_width);
	if (atomi_nums == 0)
		set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result2, p_height, p_width);
	if (atom_nums != 0 || atomi_nums != 0)
	{
		add_fun_absob << <blockSize_P, threadsPerBlock >> >(cuda_result1, cuda_result2, p_height, p_width);
		//������õ�һ�������ά����P��
		hipfftComplex* rowHead2;
		rowHead2 = (hipfftComplex *)((char *)p_PitchedPtr + pitch2 * (slice - 1));
		copy_result_to_p << <blockSize_P, threadsPerBlock >> >(cuda_result1, rowHead2, p_height, p_width,
																parameter);
	}
}


void initKernel_P(double* atom_slice, int slices, double* absorb_n, double* proj_coff_mat,
				  int p_height, int p_width,
                  double* atom_slice_i, double* absorb_n_i,
                  double* proj_coff_mat_i, double* cuda_s_2,
                  hipfftComplex* p_PitchedPtr, size_t pitch2, double* cuda_kx,
                  double* cuda_ky, double parameter, double* cuda_aperTrue, double sigma, dim3 blockSize_P
)
{

	// cuda_result1����һ�������н����������Ի�ԭ���Ե�Ԫ���ۼӽ��
	// cuda_resul2����һ�������мȴ����������ִ���ԭ���Ե�Ԫ���ۼӽ��
	hipfftComplex *cuda_result1, *cuda_result2;
	hipMalloc((void**)&cuda_result1, sizeof(hipfftComplex) * p_height * p_width);
	hipMalloc((void**)&cuda_result2, sizeof(hipfftComplex) * p_height * p_width);
	// ����cal_atomORion_fun��plan��accumulate��set_Zero����Ҫ���߳���Ŀ
	// dim3 threadsPerBlock_P(16, 16);
	// dim3 blockSize_P((p_height + threadsPerBlock.y - 1) / threadsPerBlock.y,
	//                  (p_width + threadsPerBlock.x - 1) / threadsPerBlock.x);
	// ��¼ÿһ�㴫����ɺ�proj_coff_mat��proj_coff_mat_i���±�����Ҫ���ƶ���ƫ����
	int sumSeries_n = 0;
	int sumSeries_n_i = 0;
	//����ʼ����Slices��͸�䴫��������P
	int allAtoms=0, allAtomsi=0;
	for (int slice = 1; slice <= slices; slice++)
	{
		cal_p(atom_slice, slice, absorb_n, &sumSeries_n, proj_coff_mat, blockSize_P,
		      cuda_result1, p_height, p_width,
		      atom_slice_i, absorb_n_i, proj_coff_mat_i,
		      cuda_result2,  cuda_s_2, &sumSeries_n_i,
		      p_PitchedPtr, pitch2, cuda_kx, cuda_ky,
			  parameter, &allAtoms, &allAtomsi);
	}
	// printf_2d(p_PitchedPtr, p_width, p_height, slices, pitch2);
	printf("atoms: %d, atomsi: %d\n", allAtoms, allAtomsi);
	if(cuda_result1!=NULL)
	{
		hipFree(cuda_result1);
	}
	if(cuda_result2!=NULL)
	{
		hipFree(cuda_result2);
	}
	//����P����ifftShift����Ҫ���߳���Ŀ
	// dim3 p_threadsPerBlock(32, 32);
	int p_blockX, p_blockY;
	//����P��AperTrue��ˡ�scaler��p_exp_sigma����Ҫ���߳���Ŀ
	setThreadGrid(slices, p_height * p_width, &p_blockX, &p_blockY);
	dim3 p1_dimGrid(p_blockX, p_blockY);
	p_aperTrue_pointMul << <p1_dimGrid, threadsPerBlock >> >(p_PitchedPtr, pitch2, cuda_aperTrue, p_height,
															   p_width, slices, p_height * p_width);
	setThreadGrid(slices, (p_height * p_width) / 2, &p_blockX, &p_blockY);
	dim3 p_dimGrid(p_blockX, p_blockY);
	ifftShift << <p_dimGrid, threadsPerBlock >> >(p_PitchedPtr, pitch2, p_height, p_width, slices,
	                                                p_height * p_width);
	hipfftHandle p;
	hipfftComplex* rowHead;
	
	for (int i = 0; i < slices; i++)
	{
		rowHead = (hipfftComplex *)((char *)p_PitchedPtr + i * pitch2);
		hipfftPlan2d(&p, p_height, p_width, HIPFFT_C2C);
		hipfftExecC2C(p, rowHead, rowHead, HIPFFT_BACKWARD);
		hipfftDestroy(p);
	}
	p_exp_sigma << <p1_dimGrid, threadsPerBlock >> >(p_PitchedPtr, pitch2, p_height, p_width, slices,
	                                                   p_height * p_width, sigma / 1000);
}


size_t men_check(int gpu_count)
{
	size_t gpu_free = 0;
	size_t gpu_total = 0;
	for (int i = 0; i < gpu_count; i++)
	{
		hipSetDevice(i);
		size_t free = 0;
		size_t total = 0;
		hipMemGetInfo(&free, &total);
		gpu_free += free;
		gpu_total += total;
	}
	if (gpu_free > gpu_total)
	{
		printf("Out Of Memory\n");
		return 0;
	}
	return gpu_free;
}


__global__ void accumulate_result(double* result_sum, double* result, int w_num, int aper_num, int depth,
                                  double* gfsf)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	if (id < aper_num * depth)
	{
		double temp = 0;
		for (int j = 0; j < w_num; j++)
		{
			temp += result_sum[id + j * aper_num * depth] * gfsf[j];
		}
		result[id] = temp;
	}
}





__global__ void accumulate_result2(double* result_sum, double* result, int w_num, int aper_num, int depth, int len,
                                  double* gfsf)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	if (id < aper_num * depth * len)
	{
		double temp = 0;
		for (int j = 0; j < w_num; j++)
		{
			temp += result_sum[id + j * aper_num * depth * len] * gfsf[j];
		}
		result[id] = temp;
	}
}


void myProjectfun(double* my_real_w, double* my_img_w, double* aper,
 				  double* atom_slice, double* absorb_n,double* atom_slice_i,
                  double* absorb_n_i, double* proj_coff_mat, 
				  double* proj_coff_mat_i, double* kx, double* ky,double* s_2,
                  double* my_real_const, double* my_img_const,
				  double* gfsf, int height, int width,
                  int depth, int p_height, int p_width, int slices, int step, int beginrow,
                  int begincol, int width_red, double sigma, int w_num, 
				  int aper_num, double parameter,double* aper_true,
				  int* mid_layer, int layer_len, double *aper2,
				  double* series_n_corr, double* series_n_i_corr,
				  double* ele_n_corr, double* ele_n_i_corr,
				  double* corr_info_matrix, int cim_len,
				  double* return_result,double* mid_layer_mat, double* potentialx, double* potentialy)
{
	/*
   ����˵����
		  my_real_w:��ʼ��w�����ʵ������,��ά����ά��Ϊ��height * width * w_num
		  my_imag_w:��ʼ��w������鲿����,��ά����ά��Ϊ��height * width * w_num
		  aper:ά��Ϊ��height * width * aper_num
		  atom_slice:����ÿ��͸��Ľ��������Ի�ԭ����Ԫ�صĸ���--> atom_slice[n1,n2,...,nk],ά��Ϊ��Slices
		  atom_slice_i:����ÿ��͸��ļ�������������ԭ����Ԫ�صĸ���--> atom_slice_i[n1,n2,...,nk],ά��Ϊ��Slices
		  proj_coff_mat:����ÿ��͸����������Ի�ԭ����Ԫ�ؼ������ϵ��
		  proj_coff_mat_i:����ÿ��͸���������������ԭ����Ԫ�ؼ������ϵ��
		  absorp_n:������������Ի�ԭ����Ԫ�ؿ��������Ե����ϵ��
		  absorp_n_i:������������Ի�ԭ����Ԫ�ؿ��������Ե����ϵ��
		  kx��ky:��ά����ά��Ϊp_height * p_width
		  s_2:ά��Ϊp_height * p_width
		  my_real_const��my_img_const:�ֱ�����մ��������ʵ��������鲿���󣬶�ά����ά�ȣ�height * width
		  gfsf:w�ı���ϵ����ά��Ϊ��1*w_num
		  height�����������
		  width:���������
		  depth:һ��Ҫ����ĵ���
		  Slices:͸�����
		  step:�ڴ�P�е������ƶ�����
		  beginRow��beginCol:��P����ʼ�����ֵ
		  width_red:��P�к����ƶ��Ĵ���
		  sigma:���ڼ���P = exp(i*sigma*P)
		  w_num:��ͬ�ĳ�ʼw����ĸ���
		  aper_num��aper����ĸ���
		  return_result�����淵�ؽ����ά��Ϊ��w_num * aper_num * depth
		  mid_layer_mat�������м�Ľ����ά��Ϊ��aper_num * depth * w_num *layer_len
*/

	printf("----------------CUDA START---------------\n");
	int gpu_count;
	hipGetDeviceCount(&gpu_count);
	size_t gpu_free = 0;
	// ��GPU�Ľ�,ֻ��һ��GPU

	size_t max_gpu_free = 0;
	size_t max_gpu_total = 0;
	hipDeviceProp_t prop;
	for(int i=0;i<gpu_count;i++)
    {
		hipSetDevice(i);
   	 	hipMemGetInfo( &avail, &total);
		if (avail > max_gpu_free){
			max_gpu_free = avail;
			max_gpu_total = total;
			max_gpu_index = i;
		}
	}
	hipSetDevice(max_gpu_index);
	printf("gpu count:%d, select gpu: %d\n", gpu_count, max_gpu_index);
	hipGetDeviceProperties(&prop,max_gpu_index);
	printf("name:%s\n",prop.name);
	printf("multiProcessorCount:%d\n",prop.multiProcessorCount);
	printf("maxThreadsPerBlock:%d\n",prop.maxThreadsPerBlock);
	printf("maxThreadsPerMultiProcessor:%d\n", prop.maxThreadsPerMultiProcessor);
	
	printf( "Max thread dimensions:  (%d, %d, %d)\n",
	prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	prop.maxThreadsDim[2] );
	printf( "Max grid dimensions:  (%d, %d, %d)\n",
	prop.maxGridSize[0], prop.maxGridSize[1],
	prop.maxGridSize[2] );
	
	maxThreads = 8 * prop.maxThreadsDim[1]  * prop.maxGridSize[1];
	// printf("maxThreads: %zu\n", maxThreads);

	// printf("%d\n",prop.multiProcessorCount*prop.maxThreadsPerMultiProcessor);
	int maxThread = prop.maxThreadsPerBlock;
	// ÿ��ʹ������߳���
	threadsPerBlock.x = sqrt(maxThread);
	threadsPerBlock.y = sqrt(maxThread);
	printf("max_gpu_free,max_gpu_total MB is %zu, %zu\n", max_gpu_free / 1024 / 1024, max_gpu_total / 1024 / 1024);
	printf("w_num:%d, slices:%d, aper_num:%d, layer_num:%d, depth:%d\n",w_num, slices, aper_num, layer_len, depth);
	printf("p_height:%d, p_width:%d, height:%d, width:%d\n", p_height, p_width, height, width);
	printf("mid_layer:");
	for(int i=0;i<layer_len;i++){
		printf("%d\t", mid_layer[i]);
	}
	printf("\n");
	//����cufftִ�мƻ�����p
	hipfftHandle p;
	size_t pitch1, pitch2;
	size_t pitch3, pitch4;

	//�������Դ��б���Kx��Ky����
	double *cuda_kx, *cuda_ky;
	//���Դ�������ռ�
	hipMalloc((void**)&cuda_kx, sizeof(double) * p_height * p_width);
	hipMalloc((void**)&cuda_ky, sizeof(double) * p_height * p_width);
	//���ڴ��е�kx,ky���Ƶ��Դ���
	hipMemcpy(cuda_kx, kx, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);
	hipMemcpy(cuda_ky, ky, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);

	//w_PitchedPtr������W��ɵ���ά�������Դ��е��׵�ַָ��
	//p_PitchedPtr������P��ɵ���ά�������Դ��е��׵�ַָ��
	hipfftComplex* w_PitchedPtr;
	hipfftComplex* p_PitchedPtr;
	hipMallocPitch((void**)&p_PitchedPtr, &pitch2, p_width * p_height * sizeof(hipfftComplex), slices);
	long p_size = p_width * p_height * sizeof(hipfftComplex) * slices;
	printf("p_size: %ld MB\n", p_size/1024/1024);
	//cuda_s_2���Դ������ڱ���s^2����
	double *cuda_s_2;
	//Ϊcuda_s_2��cuda_aperTrue���������Դ�
	hipMalloc((void**)&cuda_s_2, sizeof(double) * p_height * p_width);
	hipMemcpy(cuda_s_2, s_2, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);

	//cuda_aperTrue���Դ��б���aperTrue����
	double* cuda_aperTrue;
	hipMalloc((void**)&cuda_aperTrue, sizeof(double) * p_height * p_width);
	//���ڴ���cuda_aperTrue���Ƶ��Դ���
	hipMemcpy(cuda_aperTrue, aper_true, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);
	dim3 blockSize_P((p_height + threadsPerBlock.y - 1) / threadsPerBlock.y,
	                 (p_width + threadsPerBlock.x - 1) / threadsPerBlock.x);
	//����W�����ά������ÿһ����ά����ָ��
	hipfftComplex* rowHead1;
	//�����P�����ά������ÿһ����ά����ָ��
	hipfftComplex* rowHead2;
	for (int i = 0; i < slices ; i++){
		rowHead2 = (hipfftComplex *)((char *)p_PitchedPtr + pitch2 * i);
		set_Zero << < blockSize_P, threadsPerBlock >> >(rowHead2, p_height, p_width);
	}
	printf("potential start\n");
	start = clock();
	// ���в��P
	initKernel_P(atom_slice, slices, absorb_n, proj_coff_mat,
	             p_height, p_width, atom_slice_i, absorb_n_i, proj_coff_mat_i,
	             cuda_s_2,p_PitchedPtr, pitch2, cuda_kx,
	             cuda_ky, parameter, cuda_aperTrue, sigma, blockSize_P);

	stop = clock();
	duration=(double)(stop-start)/CLK_TCK;
	printf("potential finished\ntime=%.2lf s\n", duration);
	
	// hipfftComplex* potential_c = (hipfftComplex*)malloc(p_size);
	// hipMemcpy2D(potential_c, p_width * p_height * sizeof(hipfftComplex), p_PitchedPtr, pitch2,
	//              p_width * p_height * sizeof(hipfftComplex), slices, hipMemcpyDeviceToHost);
	// for(int i=0;i<p_width * p_height * slices;i++){
	// 	potentialx[i] = potential_c[i].x;
	// 	potentialy[i] = potential_c[i].y;
	// }
	// printf("X-----%.6f, Y-------%.6f\n", potentialx[0], potentialy[0]);



	if (cuda_s_2 != NULL)
	{
		hipFree(cuda_s_2);
	}
	if (cuda_kx != NULL)
	{
		hipFree(cuda_kx);
	}
	if (cuda_ky != NULL)
	{	
		hipFree(cuda_ky);
	}
	if (cuda_aperTrue != NULL)
	{
		hipFree(cuda_aperTrue);
	}	
	hipDeviceSynchronize();
	
	// ���p��ȷ

	// �����������ٿռ�
	// ��w����ֵ����Ҫ�Ŀռ�
	double *cuda_real, *cuda_img;
	hipMalloc((void**)&cuda_real, sizeof(double) * height * width * w_num);
	hipMalloc((void**)&cuda_img, sizeof(double) * height * width * w_num);
	//��W*w_num��ֵ���Ƶ��Դ���
	hipMemcpy(cuda_real, my_real_w, sizeof(double) * height * width * w_num, hipMemcpyHostToDevice);
	hipMemcpy(cuda_img, my_img_w, sizeof(double) * height * width * w_num, hipMemcpyHostToDevice);

	//constMatΪ��մ�������
	hipfftComplex* constMat = (hipfftComplex*)malloc(width * height * sizeof(hipfftComplex));
	//t_constMat���Դ��е��׵�ַָ�룬constMat���������ڴ���׵�ַָ��
	hipfftComplex* cuda_constMat;
	hipMallocPitch((void**)&cuda_constMat, &pitch4, width * sizeof(hipfftComplex), height);
	//���ⲿ��������մ������󣬸�ֵ���Դ���
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			constMat[i * width + j].x = my_real_const[i * width + j];
			constMat[i * width + j].y = my_img_const[i * width + j];
		}
	}
	hipMemcpy(cuda_constMat, constMat, sizeof(hipfftComplex) * height * width, hipMemcpyHostToDevice);
	if(constMat != NULL)
	{
		free(constMat);
	}

	//��aper�����Դ�ռ䲢��ֵ���ڴ��и��Ƶ��Դ���	
	double* aper_cuda;
	hipMalloc((void**)&aper_cuda, sizeof(double) * height * width * aper_num);
	hipMemcpy(aper_cuda, aper, sizeof(double) * height * width * aper_num, hipMemcpyHostToDevice);

	double* result_sum_dev;
	hipMalloc((void**)&result_sum_dev, sizeof(double) * aper_num * depth * w_num);
	double* result_layer_dev;
	double* aper2_cuda;

	if (layer_len > 0)
	{
		hipMalloc((void**)&result_layer_dev, sizeof(double) * aper_num * depth * w_num * layer_len);
		hipMalloc((void**)&aper2_cuda, sizeof(double) * height * width * aper_num);
		hipMemcpy(aper2_cuda, aper2, sizeof(double) * height * width * aper_num, hipMemcpyHostToDevice);
	}

	double* aper_result;
	// dim3 threadsPerBlock(32, 32);
	int VOL = height * width;
	int W_threadsize = VOL / 2;
	int blockX, blockY;

	gpu_free  = get_memory(max_gpu_index);
	int w_batchs = gpu_free / VOL / (sizeof(hipfftComplex) * w_num +  sizeof(double) * aper_num) >> 2;

	printf("GPU Free %zu MB\n", gpu_free / 1024 / 1024);
	if (gpu_free == 0)
	{
		return;
	}
	int n_cufft[2] = { width, height};
	// ��������Կ�Ӳ���ڴ棬depth��Ҫ�����ν��д���
	// int w_batchs = gpu_free / VOL / sizeof(hipfftComplex) / 6; //ÿ�μ�����ٸ��� 
	int batch_n = depth / w_batchs; // Ҫ������ٴ�

	for (int w_series = 0; w_series < w_num; w_series++)
	{
		//��������Ҫ��w_series��ͬ��w��W����ֵ��������Ҫ�ó�ÿ��w���׵�ַ
		int size = height * width * w_series;
		double* cuda_real_head = cuda_real + size;
		double* cuda_img_head = cuda_img + size;
		// ��depth��������ν�������,��w_series����������ڵڼ���λ��
		int rsd_num = w_series * aper_num * depth; //0 24 48
		
		for (int b = 0; b <= batch_n; b++)
		{
			// 20201112 ����ĵڼ���
			int layer_i = 0;
			int batch_start = b * w_batchs; //0 6 9 8 2
			int batch_end = (b + 1) * w_batchs; //3 9 12 10
			if (batch_end > depth)
				batch_end = depth; // 8 8
			// ÿ�μ���ĵ���
			int batch_size = batch_end - batch_start; //2 -1 0
			start = clock();
			if (batch_size > 0)
			{
				// �����￪�ٿռ�
				if (b == 0)
				{
					hipDeviceSynchronize();
					hipMallocPitch((void**)&w_PitchedPtr, &pitch1, VOL * sizeof(hipfftComplex), batch_size);
					hipDeviceSynchronize();
				}
				else if (b == batch_n)
				{
					hipDeviceSynchronize();
					if (w_PitchedPtr != NULL)
					{
						hipFree(w_PitchedPtr);
					}
					hipDeviceSynchronize();
					hipMallocPitch((void**)&w_PitchedPtr, &pitch1, VOL * sizeof(hipfftComplex), batch_size);
					hipDeviceSynchronize();
				}
				//�������w��p��ˡ�w����մ��������ˡ�scaler��ifft2��fft2����Ҫ���߳���Ŀ
				setThreadGrid(batch_size, VOL, &blockX, &blockY);
				dim3 blockSize(blockX, blockY);

				//������һ��ʼ��W��ֵ
				hipDeviceSynchronize();
				initKernel_W << <blockSize, threadsPerBlock >> >(w_PitchedPtr, pitch1, height, width, batch_size, VOL,
				                                                 cuda_real_head,
				                                                 cuda_img_head);
				hipDeviceSynchronize();
				// ����ÿһ��Ĵ���ֱ��w4
				for (int slice = 0; slice < slices; slice++)
				{
					// ����ÿһ��͸��p���׵�ַ
					rowHead2 = (hipfftComplex *)((char *)p_PitchedPtr + pitch2 * slice);
					// ���еĵ�ͬʱ��slice��
					hipDeviceSynchronize();
					w_p_pointMul << <blockSize, threadsPerBlock >> >(w_PitchedPtr, rowHead2, pitch1, height, width,
																	 batch_size, height * width, p_width, 
																	 beginrow, begincol, width_red, step, batch_start);
					hipDeviceSynchronize();
					rowHead1 = (hipfftComplex *)((char *)w_PitchedPtr);
					hipfftPlanMany(&p, 2, n_cufft, NULL, 1, VOL, NULL, 1, VOL, HIPFFT_C2C, batch_size);
					hipfftExecC2C(p, rowHead1, rowHead1, HIPFFT_FORWARD);
					hipDeviceSynchronize();
					hipfftDestroy(p);
					// ��data���ݸ�w_PitchedPtr
					//��w1������fft2������Ľ����Ϊw2
					//����һ����w2������մ�������(������ΪconstM)���е�˲�����Ľ����Ϊw3
					w_constM_pointMul << <blockSize, threadsPerBlock >> >(
						w_PitchedPtr, pitch1, cuda_constMat, height, width,
						batch_size, height * width);
					hipDeviceSynchronize();
					//����һ����w3����ifft2�����õ�w4

					// 20201112 modify by ypj
					// �����м����
					hipDeviceSynchronize();
					if (layer_i < layer_len && (slice + 1) == mid_layer[layer_i])
					{	
						// printf("cur_mid_layer: %d\n", mid_layer[layer_i]);
						hipMallocPitch((void**)&aper_result, &pitch3, sizeof(double) * VOL, batch_size);
						for (int i = 0; i < aper_num; i++)
						{
							//  ��ȡ��ÿ��aper���׵�ַ
							double* aper2_head = aper2_cuda + i * VOL; //�ڼ���aper
							absw_and_aperpm<< <blockSize, threadsPerBlock >> >(w_PitchedPtr, aper_result, aper2_head, pitch1, pitch3, height, width, batch_size, VOL);
							// w_final������aper������е�˱��浽result_PitchedPtr��												
							hipDeviceSynchronize();
							int cur_start = i * depth + batch_start + rsd_num + layer_i * aper_num * depth * w_num;
							double* result_layer_dev_head = result_layer_dev + cur_start;
							// ����Ԫ�����
							int h = log(VOL) / log(2); // 16
							if (pow(2, h) != VOL)
							{
								h++;
							}
							int thread_size = pow(2, h) / 2; // 2**15
							while (thread_size > 0)
							{
								int t1 = 1024;
								int t2 = thread_size / t1; // 2 ** 5 == 32
								if (thread_size / 1024 == 0)
								{
									t1 = thread_size;  // 2 ** 15
									t2 = 1; // 1
								}
								dim3 bthread(1, t1); //	(1, 2**15)
								dim3 g(batch_size, t2); // (2999, 1)
								// aper_result��aper����Ժ�ľ�����ͽ�������result_sum_dev_head��
								sum_w_kernel << < g, bthread >> >(aper_result, pitch3, VOL, thread_size
															, result_layer_dev_head, batch_size);	
								hipDeviceSynchronize();
								thread_size /= 2; // 2��15�η���16��
							}
						} // aperѭ������
						hipDeviceSynchronize();
						if (aper_result != NULL)
						{
							hipFree(aper_result);
						}
						hipDeviceSynchronize();
						layer_i += 1;
					}
					hipDeviceSynchronize();
					rowHead1 = (hipfftComplex *)((char *)w_PitchedPtr);
					hipfftPlanMany(&p, 2, n_cufft, NULL, 1, VOL, NULL, 1, VOL, HIPFFT_C2C, batch_size);
					hipfftExecC2C(p, rowHead1, rowHead1, HIPFFT_BACKWARD);
					hipDeviceSynchronize();
					hipfftDestroy(p);
					hipDeviceSynchronize();
					//��Ϊcufft��������Ƶ�ԭ����һ���ĵõ�w4�������õ�����ֵ�Ĵ�С����ʵ��ֵ��height*width??????????????��???height*width????,?????????????????
					scaler << <blockSize, threadsPerBlock >> >(w_PitchedPtr, pitch1, height, width, batch_size,
					                                           height * width);
					hipDeviceSynchronize();

				} // �ܲ����������
				//��ʼ�����벻ͬ��aper������е�˲�����Ȼ�󽫵�˺�Ľ�����о���Ԫ�ص��ۼ�
				//����W����ifftShift����Ҫ���߳���Ŀ
				int w_blockX, w_blockY;
				setThreadGrid(batch_size, W_threadsize, &w_blockX, &w_blockY);
				dim3 w_dimGrid(w_blockX, w_blockY);

				// ����fft2��fftshift
				hipDeviceSynchronize();
				rowHead1 = (hipfftComplex *)((char *)w_PitchedPtr);
				hipfftPlanMany(&p, 2, n_cufft, NULL, 1, VOL, NULL, 1, VOL, HIPFFT_C2C, batch_size);
				hipfftExecC2C(p, rowHead1, rowHead1, HIPFFT_FORWARD);
				hipDeviceSynchronize();
				hipfftDestroy(p);

				hipDeviceSynchronize();
				ifftShift << <w_dimGrid, threadsPerBlock >> >(w_PitchedPtr, pitch1, height, width, batch_size, VOL);
				hipDeviceSynchronize();
				// VOL * batch_size  ==> aper_num * depth * w_num
				hipMallocPitch((void**)&aper_result, &pitch3, VOL * sizeof(double), batch_size);
				for (int i = 0; i < aper_num; i++)
				{
					//  ��ȡ��ÿ��aper���׵�ַ
					double* aper_head = aper_cuda + i * VOL; //��һ��aper
					absw_and_aperpm<< <blockSize, threadsPerBlock >> >(w_PitchedPtr, aper_result, aper_head, pitch1, pitch3, height, width, batch_size, VOL);
					// w_final������aper������е�˱��浽result_PitchedPtr��				
					hipDeviceSynchronize();
					// ����Ԫ�����
					int h = log(VOL) / log(2); // 16
					if (pow(2, h) != VOL)
					{
						h++;
					}
					int thread_size = pow(2, h) / 2;
					// aper_num * depth * w_num
					double* result_sum_dev_head = result_sum_dev + i * depth  + rsd_num + batch_start;
					while (thread_size > 0)
					{
						int t1 = 1024;
						int t2 = thread_size / t1;
						if (thread_size / 1024 == 0)
						{
							t1 = thread_size;
							t2 = 1;
						}
						dim3 bthread(1, t1); //	(1,1024)
						dim3 g(batch_size, t2); // (3,32)
						//	aper_result��aper����Ժ�ľ�����ͽ�������result_sum_dev_head��
						sum_w_kernel << < g, bthread >> >(aper_result, pitch3, VOL, thread_size
						                            , result_sum_dev_head, batch_size);
						hipDeviceSynchronize();
						thread_size /= 2; // 2��15�η���16��
					}
					//printf_dd(result_sum_dev, aper_num * depth * w_num, i * depth  + rsd_num + batch_start);
				} // aperѭ������
				hipDeviceSynchronize();
				if (aper_result != NULL)
				{
					hipFree(aper_result);
				}
				hipDeviceSynchronize();
			}
			stop = clock();
			duration=(double)(stop-start)/CLK_TCK;
			printf("w: (%d/%d), Pixel complete: (%d/%d), time per pixel: %.2lf sec\n", w_series+1, w_num, batch_start, depth, duration/batch_size);
		} // �ֿ�ѭ������
		hipDeviceSynchronize();
		if (w_PitchedPtr != NULL)
		{
			hipFree(w_PitchedPtr);
		}
		hipDeviceSynchronize();
	} // w_numѭ������

	rowHead1 = NULL;
	rowHead2 = NULL;
	hipfftDestroy(p);
	
	if (cuda_real!=NULL)
	{
		hipFree(cuda_real);
	}
	if(cuda_img!=NULL)
	{
		hipFree(cuda_img);
	}
	if(cuda_constMat!=NULL)
	{
		hipFree(cuda_constMat);
	}
	if(p_PitchedPtr!=NULL)
	{
		hipFree(p_PitchedPtr);
	}
	if(aper_cuda!=NULL)
	{
		hipFree(aper_cuda);
	}
	if(aper2_cuda!=NULL)
	{
		hipFree(aper2_cuda);
	}

	setThreadGrid(aper_num, depth, &blockX, &blockY);
	dim3 blockSize(blockX, blockY);
	double* cuda_temp2;
	hipMalloc((void **)&cuda_temp2, sizeof(double) * aper_num * depth);
	double* cuda_gfsf;
	hipMalloc((void**)&cuda_gfsf, sizeof(double) * w_num);
	hipMemcpy(cuda_gfsf, gfsf, sizeof(double) * w_num, hipMemcpyHostToDevice);
	accumulate_result << <blockSize, threadsPerBlock >>
		>(result_sum_dev, cuda_temp2, w_num, aper_num, depth, cuda_gfsf);
	double* result_sum = (double*)malloc(sizeof(double) * aper_num * depth);
	hipMemcpy(result_sum, cuda_temp2, sizeof(double) * aper_num * depth, hipMemcpyDeviceToHost);
	for (int i = 0; i < aper_num * depth; i++)
	{
		return_result[i] = result_sum[i];
	}
	// printf("\n");
	if (cuda_temp2!=NULL)
	{
		hipFree(cuda_temp2);
	}
	if (result_sum!=NULL)
	{
		free(result_sum);
	}
	if(result_sum_dev!=NULL)
	{
		hipFree(result_sum_dev);
	}
	// 20201114 create by ypj 
	if (layer_len > 0){
		double* cuda_temp3;
		hipMalloc((void **)&cuda_temp3, sizeof(double) * aper_num * depth * layer_len);
		setThreadGrid(aper_num * layer_len, depth , &blockX, &blockY);
		dim3 blockSize(blockX, blockY);
		accumulate_result2 << <blockSize, threadsPerBlock >>
			>(result_layer_dev , cuda_temp3, 
			w_num, aper_num, depth, layer_len, cuda_gfsf);
		double* result_sum2 = (double*)malloc(sizeof(double) * aper_num * depth * layer_len);
		hipMemcpy(result_sum2, cuda_temp3, sizeof(double) * aper_num * depth * layer_len, hipMemcpyDeviceToHost);
		for (int i = 0; i < aper_num * depth * layer_len; i++)
		{
			mid_layer_mat[i] = result_sum2[i];
		}
		if (result_sum2!=NULL)
		{
			free(result_sum2);
		}
		if (cuda_temp3!=NULL)
		{
			hipFree(cuda_temp3);
		}
		if (result_layer_dev!=NULL)
		{
			hipFree(result_layer_dev);
		}
	}
	if(cuda_gfsf!=NULL)
	{
		hipFree(cuda_gfsf);
	}
	printf("----------------CUDA Finished---------------\n");
}
