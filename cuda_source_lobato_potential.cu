#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include ""
#include <time.h>
#include <math.h>

// ��ʱ
clock_t start, stop; 
double duration;
size_t avail, total;

#ifdef __linux__
 #define  CLK_TCK CLOCKS_PER_SEC
#endif

dim3 threadsPerBlock(32, 32);
size_t maxThreads;
int max_gpu_index = 0;


//��ʼ�����㺯��
__global__ void set_Zero(hipfftComplex* cuda_result, int Height, int Width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < Height && col < Width)
	{
		cuda_result[row * Width + col].x = 0;
		cuda_result[row * Width + col].y = 0;
	}
}

//��֯cal_atom_ion_part1��exp_kxX_kyY1��sum_exp_prodִ��˳����
__global__ void cal_atomORion_fun(hipfftComplex* cuda_result, float* coff, double* s_2, int atom_nums
                                  , double* kx, double* ky, int Height, int Width, int VOL)
{
	//���������Ի�ԭ����
	//cal_atom_ion_part1�����Ľ�������sum��
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atom_nums)
	{
		if (row < Height && col < Width)
		{
			float part1 = coff[atom * 14 + 3] * (2 + coff[atom * 14 + 4] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 4] * s_2[row * Width + col] * 4, 2);
			float part2 = coff[atom * 14 + 5] * (2 + coff[atom * 14 + 6] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 6] * s_2[row * Width + col] * 4, 2);
			float part3 = coff[atom * 14 + 7] * (2 + coff[atom * 14 + 8] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 8] * s_2[row * Width + col] * 4, 2);
			float part4 = coff[atom * 14 + 9] * (2 + coff[atom * 14 + 10] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 10] * s_2[row * Width + col] * 4, 2);
			float part5 = coff[atom * 14 + 11] * (2 + coff[atom * 14 + 12] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 14 + 12] * s_2[row * Width + col] * 4, 2);
			float sum = coff[atom * 14 + 2] * (part1 + part2 + part3 + part4 + part5) * expf((-1) * coff[atom * 14 + 13] * s_2[row * Width + col]);
			float temp = (-2) * (coff[atom * 14 + 0] * kx[row * Width + col] + coff[atom * 14 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + cosf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + sinf(temp) * sum;
		}
	}
}

__global__ void cal_absorb_fun(hipfftComplex* cuda_result, float* coff, float* coff_absob, double* s_2, int atom_nums
                               , double* kx, double* ky, int Height, int Width, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atom_nums)
	{
		if (row < Height && col < Width)
		{
			
			float part1 = coff_absob[atom * 10 + 0] * expf((-1) * coff_absob[atom * 10 + 1] * s_2[row * Width + col]);
			float part2 = coff_absob[atom * 10 + 2] * expf((-1) * coff_absob[atom * 10 + 3] * s_2[row * Width + col]);
			float part3 = coff_absob[atom * 10 + 4] * expf((-1) * coff_absob[atom * 10 + 5] * s_2[row * Width + col]);
			float part4 = coff_absob[atom * 10 + 6] * expf((-1) * coff_absob[atom * 10 + 7] * s_2[row * Width + col]);
			float part5 = coff_absob[atom * 10 + 8] * expf((-1) * coff_absob[atom * 10 + 9] * s_2[row * Width + col]);
			float sum = coff[atom * 14 + 2] * (part1 + part2 + part3 + part4 + part5);
			float temp = (-2) * (coff[atom * 14 + 0] * kx[row * Width + col] + coff[atom * 14 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + (-1) * sinf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + cosf(temp) * sum;
		
		}
	}
}


//һ������Ԫ��ԭ���Ի������Լ��϶�Ӧ������ֵ�����������cuda_result��
__global__ void add_fun_absob(hipfftComplex* cuda_result, hipfftComplex* cuda_absorb, int Height, int Width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < Height && col < Width)
	{
		cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + cuda_absorb[row * Width + col].x;
		cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + cuda_absorb[row * Width + col].y;
	}
}


__global__ void cal_atomAndion_fun(hipfftComplex* cuda_result,float* coff, double* s_2, int atomi_nums
                                   , double* kx, double* ky
                                   , int Height, int Width, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atomi_nums)
	{
		if (row < Height && col < Width)
		{
			float atom_part1 = coff[atom * 26 + 4] * (2 + coff[atom * 26 + 5] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 5] * s_2[row * Width + col] * 4, 2);
			float atom_part2 = coff[atom * 26 + 6] * (2 + coff[atom * 26 + 7] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 7] * s_2[row * Width + col] * 4, 2);
			float atom_part3 = coff[atom * 26 + 8] * (2 + coff[atom * 26 + 9] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 9] * s_2[row * Width + col] * 4, 2);
			float atom_part4 = coff[atom * 26 + 10] * (2 + coff[atom * 26 + 11] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 11] * s_2[row * Width + col] * 4, 2);
			float atom_part5 = coff[atom * 26 + 12] * (2 + coff[atom * 26 + 13] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 13] * s_2[row * Width + col] * 4, 2);
			float atom_sum = coff[atom * 26 + 3] * (atom_part1 + atom_part2 + atom_part3 + atom_part4 + atom_part5);

			float ion_part1 = coff[atom * 26 + 15] * (2 + coff[atom * 26 + 16] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 16] * s_2[row * Width + col] * 4, 2);
			float ion_part2 = coff[atom * 26 + 17] * (2 + coff[atom * 26 + 18] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 18] * s_2[row * Width + col] * 4, 2);
			float ion_part3 = coff[atom * 26 + 19] * (2 + coff[atom * 26 + 20] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 20] * s_2[row * Width + col] * 4, 2);
			float ion_part4 = coff[atom * 26 + 21] * (2 + coff[atom * 26 + 22] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 22] * s_2[row * Width + col] * 4, 2);
			float ion_part5 = coff[atom * 26 + 23] * (2 + coff[atom * 26 + 24] * s_2[row * Width + col] * 4) / powf(1 + coff[atom * 26 + 24] * s_2[row * Width + col] * 4, 2);
			float ion_sum = coff[atom * 26 + 14] * (ion_part1 + ion_part2 + ion_part3 + ion_part4 + ion_part5);

			float sum = coff[atom * 26 + 2] * (atom_sum + ion_sum) * expf((-1) * coff[atom * 26 + 25] * s_2[row * Width + col]);

			float temp = (-2) * (coff[atom * 26 + 0] * kx[row * Width + col] + coff[atom * 26 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + cosf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + sinf(temp) * sum;
			
		}
	}	
}

__global__ void cal_absorb_fun2(hipfftComplex* cuda_result, float* coff, float* coff_absob, double* s_2, int atomi_nums
                                , double* kx, double* ky
                                , int Height, int Width, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int atom = id / VOL;
	int row = (id % VOL) / Width;
	int col = (id % VOL) % Width;
	if(atom < atomi_nums)
	{
		if (row < Height && col < Width)
		{	
			float part1 = coff_absob[atom * 10 + 0] * expf((-1) * coff_absob[atom * 10 + 1] * s_2[row * Width + col]);
			float part2 = coff_absob[atom * 10 + 2] * expf((-1) * coff_absob[atom * 10 + 3] * s_2[row * Width + col]);
			float part3 = coff_absob[atom * 10 + 4] * expf((-1) * coff_absob[atom * 10 + 5] * s_2[row * Width + col]);
			float part4 = coff_absob[atom * 10 + 6] * expf((-1) * coff_absob[atom * 10 + 7] * s_2[row * Width + col]);
			float part5 = coff_absob[atom * 10 + 8] * expf((-1) * coff_absob[atom * 10 + 9] * s_2[row * Width + col]);
			float sum = coff[atom * 26 + 2] * (part1 + part2 + part3 + part4 + part5);
			float temp = (-2) * (coff[atom * 26 + 0] * kx[row * Width + col] + coff[atom * 26 + 1] * ky[row * Width + col]
			) * 3.14159265;
			cuda_result[row * Width + col].x = cuda_result[row * Width + col].x + (-1) * sinf(temp) * sum;
			cuda_result[row * Width + col].y = cuda_result[row * Width + col].y + cosf(temp) * sum;
		}
	}
}


__global__ void copy_result_to_p(hipfftComplex* result, hipfftComplex* P, int Height, int Width, float parameter)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < Height && col < Width)
	{
		P[row * Width + col].x = result[row * Width + col].x * parameter;
		P[row * Width + col].y = result[row * Width + col].y * parameter;
	}
}

//��P����ifftShift����
__global__ void ifftShift(hipfftComplex* P_PitchedPtr, size_t pitch, int P_Height, int P_Width, int P_Slices, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int d_id = id / (VOL / 2);
	int b_id = id % (VOL / 2) / (VOL / 4);
	int e_id = id % (VOL / 2) % (VOL / 4);

	int row = e_id / (P_Width / 2);
	int col = e_id % (P_Width / 2);

	int dest_row, dest_col;

	if (d_id < P_Slices)
	{
		if (b_id == 0)
		{
			// if(row == 0 && col == 0 && d_id == 0)
			// 	printf("invoke");
			dest_row = row + (P_Height / 2);
			dest_col = col + (P_Width / 2);
		}
		else if (b_id == 1)
		{
			col = col + (P_Width / 2);
			dest_row = row + (P_Height / 2);
			dest_col = col - (P_Width / 2);
		}
		// if(row == 0 && col == 2 && d_id == 0)
		// 	printf("(%d,%d,%d),(%d,%d),(%d,%d)\n",b_id,row,col,dest_row,dest_col,P_Height,P_Width);
		hipfftComplex* rowHead = (hipfftComplex *)((char *)P_PitchedPtr + d_id * pitch);
		float e1_real = rowHead[row * P_Width + col].x;
		float e1_img = rowHead[row * P_Width + col].y;

		float e2_real = rowHead[dest_row * P_Width + dest_col].x;
		float e2_img = rowHead[dest_row * P_Width + dest_col].y;

		rowHead[row * P_Width + col].x = e2_real;
		rowHead[row * P_Width + col].y = e2_img;

		rowHead[dest_row * P_Width + dest_col].x = e1_real;
		rowHead[dest_row * P_Width + dest_col].y = e1_img;
	}
}

//p��AperTrue������
__global__ void p_aperTrue_pointMul(hipfftComplex* P_PitchedPtr, size_t pitch, double* aperTrue, int P_Height,
                                    int P_Width, int Slices, int VOL)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int p_id = id / VOL;
	int row = (id % VOL) / P_Width;
	int col = (id % VOL) % P_Width;

	if (p_id < Slices)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)P_PitchedPtr + p_id * pitch);

		if (col < P_Width && row < P_Height)
		{
			rowHead[col + row * P_Width].x = rowHead[col + row * P_Width].x * aperTrue[col + row * P_Width];
			rowHead[col + row * P_Width].y = rowHead[col + row * P_Width].y * aperTrue[col + row * P_Width];
		}
	}
}


//����exp(i * simga * P)
__global__ void p_exp_sigma(hipfftComplex* P_PitchedPtr, size_t pitch, int P_Height, int P_Width, int Slices, int VOL,
                            float sigma)
{
	int id = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x
	);
	int p_id = id / VOL;
	int row = (id % VOL) / P_Width;
	int col = (id % VOL) % P_Width;

	if (p_id < Slices)
	{
		hipfftComplex* rowHead = (hipfftComplex *)((char *)P_PitchedPtr + p_id * pitch);

		if (col < P_Width && row < P_Height)
		{
			float real = (-1) * rowHead[col + row * P_Width].y * sigma;
			float img = rowHead[col + row * P_Width].x * sigma;

			rowHead[col + row * P_Width].x = expf(real) * cosf(img);
			rowHead[col + row * P_Width].y = expf(real) * sinf(img);
		}
	}
}


void printf_2d(hipfftComplex* w_PitchedPtr, int width, int height, int depth, size_t pitch)
{
	//w_s��������ɵ���ά������������ڴ���׵�ַָ��
	hipfftComplex* w_p = (hipfftComplex *)malloc(sizeof(hipfftComplex) * width * height * depth);
	hipMemcpy2D(w_p, width * height * sizeof(hipfftComplex), w_PitchedPtr, pitch,
	             width * height * sizeof(hipfftComplex), depth, hipMemcpyDeviceToHost);
	//printf("%.12lf + (%.12lf)i\n", w_p[127*256+127].x, w_p[127*256+127].y);
	int k = 0;
	//for (int i = 496 * 235 + 273; i < 496 * 235 + 273 + 5; i++)
	for (int i = 0; i < 5; i++)
	{
		printf("%d================%.12lf + (%.12lf)i \n", i, w_p[i].x, w_p[i].y);
		k++;
		if ((k % (height * width)) == 0)
		{
			printf("\n=======================\n");
			printf("\n");
		}
	}
	if (w_p!=NULL)
	{
		free(w_p);
	}
}

void printf_1d(hipfftComplex* cuda_result, int height, int width)
{
	hipfftComplex* test = (hipfftComplex *)malloc(sizeof(hipfftComplex) * height * width);
	hipMemcpy(test, cuda_result, height * sizeof(hipfftComplex) * width, hipMemcpyDeviceToHost);
	int k = 0;
	for (int i = 0; i < height * width; i++)
	{
		k++;
		printf("%lf + (%lf)i ", test[i].x, test[i].y);
		if (k % width == 0)
		{
			printf("\n");
		}
	}
	if (test!=NULL)
	{
		free(test);
	}
}

void printf_dd(double* cuda_result, int start, int len)
{
	double* test = (double *)malloc(sizeof(double) * len);
	hipMemcpy(test, cuda_result+start, sizeof(double) * len, hipMemcpyDeviceToHost);
	for (int i = 0; i < len; i++)
	{
		if (fabs(test[i]-0) > 1E-6){
			printf("printf_dd[%d]: %lf\n", start+i, test[i]);
		}
	}
	if (test!=NULL)
	{
		free(test);
	}
}



void setThreadGrid(int nums, int VOL, int* blockX, int* blockY)
{
	int blockThread = threadsPerBlock.x * threadsPerBlock.y;
	int blockSum = (nums * VOL  + blockThread - 1) / blockThread;
	if (blockSum < blockThread)
	{
		*blockX = 1;
		*blockY = blockSum;
	}
	else
	{
		*blockX = (blockSum + blockThread - 1) / blockThread;
		*blockY = (blockSum + *blockX - 1) / *blockX;
	}
}

size_t get_memory(int gpu_index)
{
	size_t gpu_free = 0;
	size_t gpu_total = 0;
	hipSetDevice(gpu_index);
	hipMemGetInfo(&gpu_free, &gpu_total);
	return gpu_free;
}

// �������е�p
void cal_p(double* atom_slice, int slice, double* absorb_n, int* sumSeries_n, double* proj_coff_mat, dim3 blockSize_P,
            hipfftComplex* cuda_result1, int p_height, int p_width,
           double* atom_slice_i, double* absorb_n_i, double* proj_coff_mat_i,
           hipfftComplex* cuda_result2, double* cuda_s_2, int* sumSeries_n_i,
           hipfftComplex* p_PitchedPtr, size_t pitch2, double* cuda_kx, double* cuda_ky,
           double parameter, int *allAtoms, int *allAtomsi)
{
	//������������Ժ�ԭ���Ծ�������Ҫ�Ĳ�������
	float* coffMat;
	float* coffAbsob;
	float* coffMat_cuda;
	float* coffAbsob_cuda;

	//cuda_result_absorb���濼������
	hipfftComplex* cuda_result_absorb;
	//һ���н��������Ի�ԭ����Ԫ�صĵ���
	// slice��1��ʼ
	int atom_nums = atom_slice[slice - 1];

	size_t ava = get_memory(max_gpu_index);
	printf("%d-------%d-------%zd MB\n",slice, atom_nums, ava/1024/1024);
	*allAtoms += atom_nums;
	int p_vol = p_width * p_height;
	int x, y;

	int p_batchs = maxThreads/p_vol;
	// printf("p_batchs: %d\n", p_batchs);
	if (atom_nums > 0)
	{
		int ci = atom_nums / p_batchs + 1;
		if (absorb_n == NULL)
		{
			int len = 14 * atom_nums;
			coffMat = (float *)malloc(sizeof(float) * len);
			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atom_nums; m++)
			{
				for (int n = 0; n < 14; n++)
					coffMat[m * 14 + n] = proj_coff_mat[*sumSeries_n * 14 + m * 14 + n];
			}
			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len, hipMemcpyHostToDevice);
			set_Zero << < blockSize_P, threadsPerBlock >> >(cuda_result1, p_height, p_width);
			//�����Ѵ��ڴ渴�Ƶ��Դ��У����ͷ����ڴ��п��ٵ��ڴ档
			if (coffMat!=NULL)
			{
				free(coffMat);
			}
			
			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atom_nums)
				{
					end_atom = atom_nums;
				}
				int batch_atom = end_atom - start_atom;
				// printf("start_atom:%d end_atom:%d batch_atom:%d\n",start_atom, end_atom, batch_atom);
				if(batch_atom > 0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
					cal_atomORion_fun << < grid, threadsPerBlock >> >(cuda_result1, coffMat_cuda+start_atom*14, cuda_s_2, batch_atom
																			, cuda_kx, cuda_ky, p_height, p_width, p_vol);
					hipDeviceSynchronize();
				}
			}
		}
		else
		{
			hipMalloc((void**)&cuda_result_absorb, sizeof(hipfftComplex) * p_height * p_width);
			//����ò�ϵ������Ҫ�Ŀռ䳤��
			int len1 = 14 * atom_nums;
			int len2 = 10 * atom_nums;
			coffMat = (float *)malloc(sizeof(float) * len1);
			coffAbsob = (float *)malloc(sizeof(float) * len2);

			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atom_nums; m++)
			{
				for (int n = 0; n < 14; n++)
					coffMat[m * 14 + n] = proj_coff_mat[*sumSeries_n * 14 + m * 14 + n];
				for (int n = 0; n < 10; n++)
					coffAbsob[m * 10 + n] = absorb_n[*sumSeries_n * 10 + m * 10 + n];
			}
			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len1);
			hipMalloc((void**)&coffAbsob_cuda, sizeof(float) * len2);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len1, hipMemcpyHostToDevice);
			hipMemcpy(coffAbsob_cuda, coffAbsob, sizeof(float) * len2, hipMemcpyHostToDevice);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result1, p_height, p_width);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result_absorb, p_height, p_width);
			//�����Ѵ��ڴ渴�Ƶ��Դ��У����ͷ����ڴ��п��ٵ��ڴ档
			if (coffMat!=NULL)
			{
				free(coffMat);
			}
			if (coffAbsob!=NULL)
			{
				free(coffAbsob);
			}
			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atom_nums)
				{
					end_atom = atom_nums;
				}
				int batch_atom = end_atom - start_atom;
				if(batch_atom>0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
					// ���е�silce��͸��ʱ��cal_atomORion_fun����ĳ��Ԫ�ص� occupy_rate * ��i(1-5)Ai*exp(-Bi * s^2).* exp(-(kx*X + ky*Y)*i)
					// ���㲻���յ�һ����
					cal_atomORion_fun << <grid, threadsPerBlock >> >(cuda_result1, coffMat_cuda + start_atom * 14, cuda_s_2, batch_atom
																				, cuda_kx, cuda_ky, p_height, p_width, p_vol);
																				hipDeviceSynchronize();
																				//cal_absorb_fun ���� occupy_rate * ��i(1-5)j * Ci * exp(-Di * s^2).* exp(-(kx*X + ky*Y)*i)
					cal_absorb_fun << <grid, threadsPerBlock >> >(cuda_result_absorb, coffMat_cuda + start_atom * 14, coffAbsob_cuda + start_atom * 10
						, cuda_s_2, batch_atom, cuda_kx, cuda_ky, p_height, p_width, p_vol);
						hipDeviceSynchronize();
				}
			}		
		}
	}
	//�������գ���������������������
	if ((absorb_n != NULL) && (atom_nums != 0))
	{
		add_fun_absob << <blockSize_P, threadsPerBlock >> >(cuda_result1, cuda_result_absorb, p_height, p_width);
		hipDeviceSynchronize();
		//�������գ����ͷ���Ӧ���Դ�
		if (cuda_result_absorb!=NULL)
		{
			hipFree(cuda_result_absorb);
		}
		
	}
	//�ͷŵ�ǰ�����Դ濪�ٵĿռ�
				
	if(coffMat_cuda!=NULL)
	{
		hipFree(coffMat_cuda);
	}
	if (coffAbsob_cuda!=NULL)
	{
		hipFree(coffAbsob_cuda);
	}
	//�ۼ�һ����ƫ��ֵ
	*sumSeries_n += atom_nums;
	int atomi_nums = atom_slice_i[slice - 1];
	
	allAtomsi += atomi_nums;
	//һ�������м���ԭ��������������Ԫ�صĵ���
	if (atomi_nums > 0)
	{
		int ci = atomi_nums / p_batchs + 1;
		if (absorb_n_i == NULL)
		{
			//��proj_coff_mat_i�л�ȡ����������Ҫ������ϵ��
			//����ò�ϵ������Ҫ�Ŀռ䳤��
			int len = 26 * atomi_nums;
			coffMat = (float *)malloc(sizeof(float) * len);
			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atomi_nums; m++)
			{
				for (int n = 0; n < 26; n++)
					coffMat[m * 26 + n] = proj_coff_mat_i[*sumSeries_n_i * 26 + m * 26 + n];
			}
			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len, hipMemcpyHostToDevice);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result2, p_height, p_width);
			if (coffMat!=NULL)
			{
				free(coffMat);
			}

			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atomi_nums)
				{
					end_atom = atomi_nums;
				}
				int batch_atom = end_atom - start_atom;
				if(batch_atom>0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
					cal_atomAndion_fun << <grid, threadsPerBlock >> >(cuda_result2, coffMat_cuda + start_atom * 26, cuda_s_2, batch_atom
																		, cuda_kx, cuda_ky, p_height, p_width, p_vol);
																		hipDeviceSynchronize();
				}
			}
		}
		else
		{
			//��proj_coff_mat��absorp�л�ȡ�����м�������Ҫ������ϵ��
			hipMalloc((void**)&cuda_result_absorb, sizeof(hipfftComplex) * p_height * p_width);

			//����ò�ϵ������Ҫ�Ŀռ䳤��
			int len1 = 26 * atomi_nums;
			int len2 = 10 * atomi_nums;

			coffMat = (float *)malloc(sizeof(float) * len1);
			coffAbsob = (float *)malloc(sizeof(float) * len2);

			//һ�λ�ȡ����ǰ���������ԭ���Ժ������Ծ�������в���
			for (int m = 0; m < atomi_nums; m++)
			{
				for (int n = 0; n < 26; n++)
					coffMat[m * 26 + n] = proj_coff_mat_i[*sumSeries_n_i * 26 + m * 26 + n];
				for (int n = 0; n < 10; n++)
					coffAbsob[m * 10 + n] = absorb_n_i[*sumSeries_n_i * 10 + m * 10 + n];
			}

			hipMalloc((void**)&coffMat_cuda, sizeof(float) * len1);
			hipMalloc((void**)&coffAbsob_cuda, sizeof(float) * len2);
			hipMemcpy(coffMat_cuda, coffMat, sizeof(float) * len1, hipMemcpyHostToDevice);
			hipMemcpy(coffAbsob_cuda, coffAbsob, sizeof(float) * len2, hipMemcpyHostToDevice);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result2, p_height, p_width);
			set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result_absorb, p_height, p_width);
			//�����Ѵ��ڴ渴�Ƶ��Դ��У����ͷ����ڴ��п��ٵ��ڴ档
			if (coffMat!=NULL)
			{
				free(coffMat);
			}
			if(coffAbsob!=NULL)
			{
				free(coffAbsob);
			}

			for(int i = 0;i < ci; i++)
			{
				int start_atom = i * p_batchs;
				int end_atom = (i+1) * p_batchs;
				if(end_atom > atomi_nums)
				{
					end_atom = atomi_nums;
				}
				int batch_atom = end_atom - start_atom;
				if(batch_atom>0)
				{
					setThreadGrid(batch_atom, p_vol, &x, &y);
					dim3 grid(x, y);
				
					cal_atomAndion_fun << <grid, threadsPerBlock >> >(
							cuda_result2, coffMat_cuda + start_atom * 26, cuda_s_2, batch_atom
						, cuda_kx, cuda_ky
						, p_height, p_width, p_vol);
						hipDeviceSynchronize();
					cal_absorb_fun2 << <grid, threadsPerBlock >> >(
						cuda_result_absorb, coffMat_cuda + start_atom * 26, coffAbsob_cuda + start_atom * 10, cuda_s_2, batch_atom
						, cuda_kx, cuda_ky
						, p_height, p_width, p_vol);	
						hipDeviceSynchronize();
				}
			}
		}
	}
	
	//�����������
	//��������������ԭ���Ե�Ԫ���������������
	if ((absorb_n_i != NULL) && (atomi_nums != 0))
	{
		add_fun_absob << <blockSize_P, threadsPerBlock >> >(cuda_result2, cuda_result_absorb, p_height,
		                                                      p_width);
		//�������ղ��ͷ���Ӧ���Դ�
		if(cuda_result_absorb!=NULL)
		{
			hipFree(cuda_result_absorb);
		}
		
	}
	//�ͷŵ�ǰ�����Դ濪�ٵĿռ�
	if (coffMat_cuda!=NULL)
	{
		hipFree(coffMat_cuda);
	}
	if(coffAbsob_cuda!=NULL)
	{
		hipFree(coffAbsob_cuda);
	}
	//�ۼƵ�ǰ����ƫ����
	*sumSeries_n_i += atomi_nums;

	//��һ�����н������ӻ�ԭ���Ը�������������ԭ���Ե�Ԫ�����
	if (atom_nums == 0)
		set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result1, p_height, p_width);
	if (atomi_nums == 0)
		set_Zero << <blockSize_P, threadsPerBlock >> >(cuda_result2, p_height, p_width);
	if (atom_nums != 0 || atomi_nums != 0)
	{
		add_fun_absob << <blockSize_P, threadsPerBlock >> >(cuda_result1, cuda_result2, p_height, p_width);
		//������õ�һ�������ά����P��
		hipfftComplex* rowHead2;
		rowHead2 = (hipfftComplex *)((char *)p_PitchedPtr + pitch2 * (slice - 1));
		copy_result_to_p << <blockSize_P, threadsPerBlock >> >(cuda_result1, rowHead2, p_height, p_width,
																parameter);
	}
}


void initKernel_P(double* atom_slice, int slices, double* absorb_n, double* proj_coff_mat,
				  int p_height, int p_width,
                  double* atom_slice_i, double* absorb_n_i,
                  double* proj_coff_mat_i, double* cuda_s_2,
                  hipfftComplex* p_PitchedPtr, size_t pitch2, double* cuda_kx,
                  double* cuda_ky, double parameter, double* cuda_aperTrue, double sigma, dim3 blockSize_P
)
{

	// cuda_result1����һ�������н����������Ի�ԭ���Ե�Ԫ���ۼӽ��
	// cuda_resul2����һ�������мȴ����������ִ���ԭ���Ե�Ԫ���ۼӽ��
	hipfftComplex *cuda_result1, *cuda_result2;
	hipMalloc((void**)&cuda_result1, sizeof(hipfftComplex) * p_height * p_width);
	hipMalloc((void**)&cuda_result2, sizeof(hipfftComplex) * p_height * p_width);
	// ����cal_atomORion_fun��plan��accumulate��set_Zero����Ҫ���߳���Ŀ
	// dim3 threadsPerBlock_P(16, 16);
	// dim3 blockSize_P((p_height + threadsPerBlock.y - 1) / threadsPerBlock.y,
	//                  (p_width + threadsPerBlock.x - 1) / threadsPerBlock.x);
	// ��¼ÿһ�㴫����ɺ�proj_coff_mat��proj_coff_mat_i���±�����Ҫ���ƶ���ƫ����
	int sumSeries_n = 0;
	int sumSeries_n_i = 0;
	//����ʼ����Slices��͸�䴫��������P
	int allAtoms=0, allAtomsi=0;
	for (int slice = 1; slice <= slices; slice++)
	{
		cal_p(atom_slice, slice, absorb_n, &sumSeries_n, proj_coff_mat, blockSize_P,
		      cuda_result1, p_height, p_width,
		      atom_slice_i, absorb_n_i, proj_coff_mat_i,
		      cuda_result2,  cuda_s_2, &sumSeries_n_i,
		      p_PitchedPtr, pitch2, cuda_kx, cuda_ky,
			  parameter, &allAtoms, &allAtomsi);
	}
	// printf_2d(p_PitchedPtr, p_width, p_height, slices, pitch2);
	printf("atoms: %d, atomsi: %d\n", allAtoms, allAtomsi);
	if(cuda_result1!=NULL)
	{
		hipFree(cuda_result1);
	}
	if(cuda_result2!=NULL)
	{
		hipFree(cuda_result2);
	}
	//����P����ifftShift����Ҫ���߳���Ŀ
	// dim3 p_threadsPerBlock(32, 32);
	int p_blockX, p_blockY;
	//����P��AperTrue��ˡ�scaler��p_exp_sigma����Ҫ���߳���Ŀ
	setThreadGrid(slices, p_height * p_width, &p_blockX, &p_blockY);
	dim3 p1_dimGrid(p_blockX, p_blockY);
	p_aperTrue_pointMul << <p1_dimGrid, threadsPerBlock >> >(p_PitchedPtr, pitch2, cuda_aperTrue, p_height,
															   p_width, slices, p_height * p_width);
	setThreadGrid(slices, (p_height * p_width) / 2, &p_blockX, &p_blockY);
	dim3 p_dimGrid(p_blockX, p_blockY);
	ifftShift << <p_dimGrid, threadsPerBlock >> >(p_PitchedPtr, pitch2, p_height, p_width, slices,
	                                                p_height * p_width);
	hipfftHandle p;
	hipfftComplex* rowHead;
	
	for (int i = 0; i < slices; i++)
	{
		rowHead = (hipfftComplex *)((char *)p_PitchedPtr + i * pitch2);
		hipfftPlan2d(&p, p_height, p_width, HIPFFT_C2C);
		hipfftExecC2C(p, rowHead, rowHead, HIPFFT_BACKWARD);
		hipfftDestroy(p);
	}
	p_exp_sigma << <p1_dimGrid, threadsPerBlock >> >(p_PitchedPtr, pitch2, p_height, p_width, slices,
	                                                   p_height * p_width, sigma / 1000);
}




void myProjectfun(double* my_real_w, double* my_img_w, double* aper,
 				  double* atom_slice, double* absorb_n,double* atom_slice_i,
                  double* absorb_n_i, double* proj_coff_mat, 
				  double* proj_coff_mat_i, double* kx, double* ky,double* s_2,
                  double* my_real_const, double* my_img_const,
				  double* gfsf, int height, int width,
                  int depth, int p_height, int p_width, int slices, int step, int beginrow,
                  int begincol, int width_red, double sigma, int w_num, 
				  int aper_num, double parameter,double* aper_true,
				  int* mid_layer, int layer_len, double *aper2,
				  double* series_n_corr, double* series_n_i_corr,
				  double* ele_n_corr, double* ele_n_i_corr,
				  double* corr_info_matrix, int cim_len,
				  double* return_result,double* mid_layer_mat, double* potentialx, double* potentialy)
{
	/*
   ����˵����
		  my_real_w:��ʼ��w�����ʵ������,��ά����ά��Ϊ��height * width * w_num
		  my_imag_w:��ʼ��w������鲿����,��ά����ά��Ϊ��height * width * w_num
		  aper:ά��Ϊ��height * width * aper_num
		  atom_slice:����ÿ��͸��Ľ��������Ի�ԭ����Ԫ�صĸ���--> atom_slice[n1,n2,...,nk],ά��Ϊ��Slices
		  atom_slice_i:����ÿ��͸��ļ�������������ԭ����Ԫ�صĸ���--> atom_slice_i[n1,n2,...,nk],ά��Ϊ��Slices
		  proj_coff_mat:����ÿ��͸����������Ի�ԭ����Ԫ�ؼ������ϵ��
		  proj_coff_mat_i:����ÿ��͸���������������ԭ����Ԫ�ؼ������ϵ��
		  absorp_n:������������Ի�ԭ����Ԫ�ؿ��������Ե����ϵ��
		  absorp_n_i:������������Ի�ԭ����Ԫ�ؿ��������Ե����ϵ��
		  kx��ky:��ά����ά��Ϊp_height * p_width
		  s_2:ά��Ϊp_height * p_width
		  my_real_const��my_img_const:�ֱ�����մ��������ʵ��������鲿���󣬶�ά����ά�ȣ�height * width
		  gfsf:w�ı���ϵ����ά��Ϊ��1*w_num
		  height�����������
		  width:���������
		  depth:һ��Ҫ����ĵ���
		  Slices:͸�����
		  step:�ڴ�P�е������ƶ�����
		  beginRow��beginCol:��P����ʼ�����ֵ
		  width_red:��P�к����ƶ��Ĵ���
		  sigma:���ڼ���P = exp(i*sigma*P)
		  w_num:��ͬ�ĳ�ʼw����ĸ���
		  aper_num��aper����ĸ���
		  return_result�����淵�ؽ����ά��Ϊ��w_num * aper_num * depth
		  mid_layer_mat�������м�Ľ����ά��Ϊ��aper_num * depth * w_num *layer_len
*/

	printf("----------------CUDA START---------------\n");
	int gpu_count;
	hipGetDeviceCount(&gpu_count);
	// size_t gpu_free = 0;
	// ��GPU�Ľ�,ֻ��һ��GPU

	size_t max_gpu_free = 0;
	size_t max_gpu_total = 0;
	hipDeviceProp_t prop;
	for(int i=0;i<gpu_count;i++)
    {
		hipSetDevice(i);
   	 	hipMemGetInfo( &avail, &total);
		if (avail > max_gpu_free){
			max_gpu_free = avail;
			max_gpu_total = total;
			max_gpu_index = i;
		}
	}
	hipSetDevice(max_gpu_index);
	printf("gpu count:%d, select gpu: %d\n", gpu_count, max_gpu_index);
	hipGetDeviceProperties(&prop,max_gpu_index);
	printf("name:%s\n",prop.name);
	printf("multiProcessorCount:%d\n",prop.multiProcessorCount);
	printf("maxThreadsPerBlock:%d\n",prop.maxThreadsPerBlock);
	printf("maxThreadsPerMultiProcessor:%d\n", prop.maxThreadsPerMultiProcessor);
	
	printf( "Max thread dimensions:  (%d, %d, %d)\n",
	prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	prop.maxThreadsDim[2] );
	printf( "Max grid dimensions:  (%d, %d, %d)\n",
	prop.maxGridSize[0], prop.maxGridSize[1],
	prop.maxGridSize[2] );
	
	maxThreads = 8 * prop.maxThreadsDim[1]  * prop.maxGridSize[1];
	// printf("maxThreads: %zu\n", maxThreads);

	// printf("%d\n",prop.multiProcessorCount*prop.maxThreadsPerMultiProcessor);
	int maxThread = prop.maxThreadsPerBlock;
	// ÿ��ʹ������߳���
	threadsPerBlock.x = sqrt(maxThread);
	threadsPerBlock.y = sqrt(maxThread);
	printf("max_gpu_free,max_gpu_total MB is %zu, %zu\n", max_gpu_free / 1024 / 1024, max_gpu_total / 1024 / 1024);
	printf("w_num:%d, slices:%d, aper_num:%d, layer_num:%d, depth:%d\n",w_num, slices, aper_num, layer_len, depth);
	printf("p_height:%d, p_width:%d, height:%d, width:%d\n", p_height, p_width, height, width);
	printf("mid_layer:");
	for(int i=0;i<layer_len;i++){
		printf("%d\t", mid_layer[i]);
	}
	printf("\n");
	//����cufftִ�мƻ�����p
	// hipfftHandle p;
	size_t pitch2;

	//�������Դ��б���Kx��Ky����
	double *cuda_kx, *cuda_ky;
	//���Դ�������ռ�
	hipMalloc((void**)&cuda_kx, sizeof(double) * p_height * p_width);
	hipMalloc((void**)&cuda_ky, sizeof(double) * p_height * p_width);
	//���ڴ��е�kx,ky���Ƶ��Դ���
	hipMemcpy(cuda_kx, kx, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);
	hipMemcpy(cuda_ky, ky, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);

	//w_PitchedPtr������W��ɵ���ά�������Դ��е��׵�ַָ��
	//p_PitchedPtr������P��ɵ���ά�������Դ��е��׵�ַָ��
	// hipfftComplex* w_PitchedPtr;
	hipfftComplex* p_PitchedPtr;
	hipMallocPitch((void**)&p_PitchedPtr, &pitch2, p_width * p_height * sizeof(hipfftComplex), slices);
	long p_size = p_width * p_height * sizeof(hipfftComplex) * slices;
	printf("p_size: %ld MB\n", p_size/1024/1024);
	//cuda_s_2���Դ������ڱ���s^2����
	double *cuda_s_2;
	//Ϊcuda_s_2��cuda_aperTrue���������Դ�
	hipMalloc((void**)&cuda_s_2, sizeof(double) * p_height * p_width);
	hipMemcpy(cuda_s_2, s_2, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);

	//cuda_aperTrue���Դ��б���aperTrue����
	double* cuda_aperTrue;
	hipMalloc((void**)&cuda_aperTrue, sizeof(double) * p_height * p_width);
	//���ڴ���cuda_aperTrue���Ƶ��Դ���
	hipMemcpy(cuda_aperTrue, aper_true, sizeof(double) * p_height * p_width, hipMemcpyHostToDevice);

	dim3 blockSize_P((p_height + threadsPerBlock.y - 1) / threadsPerBlock.y,
	                 (p_width + threadsPerBlock.x - 1) / threadsPerBlock.x);
	//����W�����ά������ÿһ����ά����ָ��
	// hipfftComplex* rowHead1;
	//�����P�����ά������ÿһ����ά����ָ��
	hipfftComplex* rowHead2;
	for (int i = 0; i < slices ; i++){
		rowHead2 = (hipfftComplex *)((char *)p_PitchedPtr + pitch2 * i);
		set_Zero << < blockSize_P, threadsPerBlock >> >(rowHead2, p_height, p_width);
	}

	printf("potential start\n");
	start = clock();
	// ���в��P
	initKernel_P(atom_slice, slices, absorb_n, proj_coff_mat,
	             p_height, p_width, atom_slice_i, absorb_n_i, proj_coff_mat_i,
	             cuda_s_2,p_PitchedPtr, pitch2, cuda_kx,
	             cuda_ky, parameter, cuda_aperTrue, sigma, blockSize_P);

	stop = clock();
	duration=(double)(stop-start)/CLK_TCK;
	printf("potential finished\ntime=%.2lf s\n", duration);
	
	hipfftComplex* potential_c = (hipfftComplex*)malloc(p_size);
	hipMemcpy2D(potential_c, p_width * p_height * sizeof(hipfftComplex), p_PitchedPtr, pitch2,
	             p_width * p_height * sizeof(hipfftComplex), slices, hipMemcpyDeviceToHost);
	for(int i=0;i<p_width * p_height * slices;i++){
		potentialx[i] = potential_c[i].x;
		potentialy[i] = potential_c[i].y;
	}
	// printf("X-----%.6f, Y-------%.6f\n", potentialx[0], potentialy[0]);

	if (potential_c != NULL){
		free(potential_c);
	}
	if (cuda_s_2 != NULL)
	{
		hipFree(cuda_s_2);
	}
	if (cuda_kx != NULL)
	{
		hipFree(cuda_kx);
	}
	if (cuda_ky != NULL)
	{	
		hipFree(cuda_ky);
	}
	if (cuda_aperTrue != NULL)
	{
		hipFree(cuda_aperTrue);
	}	
	hipDeviceSynchronize();
	
	printf("----------------CUDA Finished---------------\n");
}
